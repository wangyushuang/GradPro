#include "hip/hip_runtime.h"
// ------------------ͷ�ļ�------------------
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "hipfft/hipfftw.h"
#include<complex>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <memory.h>
#include<malloc.h>
#include<iostream>
using namespace std;
#include <windows.h>
#include <process.h>

//131072��ԭʼ�����˲��õ�65536���˲���ͨ�����ݣ�50%�ص�
//65536���˲���ͨ�����ݽ������õ�16384��ͨ������
//��16384��Ƶ�����γɣ���Ŀ�귽��
//��32768��ʱ�����γɣ��õ�16384����ٲ���
//ʱ�򽵲�����������ۻ�����Ϊ32768�㣬50%�ص�

// -----------------�궨��-----------------------
//#define		ARNUM                    96       // ��Ԫ���� 
//#define     SAMPNUM                  16384    // �����γ�֡��
//#define     BeamformNumber           97       // ������
#define     NFFT					 16384	  // FFT�ĵ���
#define     PI				         3.1415926f
#define     UWC						 1500.0f     //����
#define     FS						 100000    // ����Ƶ��
typedef     float2					 Complex;
//#define     blockNum                 97
#define     threadsPerBlock          512
#define     d                        0.07f
#define     FL                       100.0f
#define     FH                       4000.0f
#define     TL                       17
#define     CHANNUM                  16
#define     FRAMELEN                 65536
#define     DOWNSAMPLE               4
#define     FIRORDER                 2048
#define     FILTER_FRAME             (2*FRAMELEN)
#define     BEAMNUM                  91
#define     THREADNUMPERBLK          256
#define     ARRAYNUM                 15
#define     STARTBEAM                15
#define     ENDBEAM                  75
#define     MAXTRACETARNUM           3
#define     M                        3
#define     PSD_LEN                  20
#define     PSD_AVG_NUM              8
#define     EPS                      1e-8
#define     SMOOTH_N                 100
#define     LINE_NUM                 16
#define     DEM_RST_LEN              1024
#define     VECTOR_P_IDX             22
#define     VECTOR_X_IDX             16
#define     VECTOR_Y_IDX             18
// --------------��������--------------------------
void Run(LPVOID lParam);
void ReadBoard1Data(LPVOID lParam);
void ReadBoard2Data(LPVOID lParam);
void DataFormatting(LPVOID lParam);
void ArraySignalProcessing(LPVOID lParam);
//---------------�¼��йصı���--------------------
HANDLE g_hReadBoard1ThreadReadyEnvent;
HANDLE g_hReadBoard2ThreadReadyEnvent;
HANDLE g_hFrameDataReadyEnvent;
//---------------��ȡ���ݱ���---------------------
int *DataBufA_B1 = NULL;
int *DataBufB_B1 = NULL;
int *DataBufA_B2 = NULL;
int *DataBufB_B2 = NULL;
float *ChannDataBufA = NULL;
float *ChannDataBufB = NULL;
float *DownSamplingDataBufA = NULL;
float *DownSamplingDataBufB = NULL;
//------------------�˲�����������-----------------
int fir1(int n,int band,float fl,float fh,float fs,int wn, float *h);
float window(int type,int n,int i,float beta);
float kaiser(int i,int n,float beta);
float bessel0(float x);
void findpeak(float *data, int *p,int dn);
void findvalley(float *data, int *p,int dn);
bool peakdetection(int beamidx,float *be,int *valley,float threshold);
void rbub(float *p,int *idx,int n);
void MySmooth(float *datain,int nDataLen,float *paraA,int nParaLen,int nOrder,int nWindow,int nStep,float *dataout);
void CalSmoothPara(float *para);

//�����׷���
float fSmoothA[4][SMOOTH_N]={0.0};                //��������������ʽ���ʱ���ù淶��������
float fPlineInfo[MAXTRACETARNUM][LINE_NUM][4]={0};//��������Ϣ
float fDlineInfo[MAXTRACETARNUM][LINE_NUM][2]={0};//�������Ϣ
int   nPlineNum = 0;
int   nDlineNum = 0;
int   nVectorPlineNum = 0;
float fVectorPlineInfo[LINE_NUM][4]={0};          //��������Ϣ
//����׷���
int   DemFreqBandNum=0;                           //����׷�����Ƶ������Ĭ������10��Ƶ��
float DemStartFreq[10]={0.0};                     //����׷�����Ƶ����ʼƵ��
float DemEndFreq[10]={0.0};                       //����׷�����Ƶ������Ƶ��
// -----------------������-------------------------
int main(int argc, char **argv)
{
	 g_hReadBoard1ThreadReadyEnvent = CreateEvent(NULL,FALSE,FALSE,NULL);
	 g_hReadBoard2ThreadReadyEnvent = CreateEvent(NULL,FALSE,FALSE,NULL);
	 g_hFrameDataReadyEnvent = CreateEvent(NULL,FALSE,FALSE,NULL);

	 //_beginthread(Run,0,NULL);	
	 _beginthread(ArraySignalProcessing,0,NULL);
	 _beginthread(DataFormatting,0,NULL);
	 _beginthread(ReadBoard1Data,0,NULL);
	 _beginthread(ReadBoard2Data,0,NULL);

     Sleep(1000);
	 getchar();
	 getchar();
     return 0;
}

__global__ void PhiShiftFactorGen(hipfftComplex *XNSS)
{
	int bid = 0,tid = 0;
	float tt = 0.0f;
	float angle=0.0f;
	float det[ARRAYNUM];
	float MovePoints[ARRAYNUM];

	bid = blockIdx.x;
	tid = threadIdx.x;
	angle=float(tid*PI/(BEAMNUM-1));

	for(int i=0;i<ARRAYNUM;i++)
	{		
	   det[i]=i*d*cos(angle)/UWC;
	   MovePoints[i]=det[i]*FS/DOWNSAMPLE;
	   tt=MovePoints[i]*2*PI*bid/NFFT;
	   XNSS[tid*ARRAYNUM*NFFT/2+i*NFFT/2+bid].x = cos(tt);
	   XNSS[tid*ARRAYNUM*NFFT/2+i*NFFT/2+bid].y = sin(tt);
	}
}

//__global__ void f_beamform(hipfftComplex *dev_fft,hipfftReal *dev_energy,int *FreqBin,hipfftComplex *PhiArray)
//{
//	__shared__ float Mabs[NFFT/2];
//	float tempX=0.0f;
//	float tempY=0.0f;
//	int nfl = 0;
//	int nfh = 0;
//	int freqbinnum = 0;
//	hipComplex XNSS;
//	hipComplex XFFTafterPinYi;
//	float ax = 0.0f,ay=0.0f,bx=0.0f,by=0.0f;
//	float energyEachBoShu = 0.0f;
//	int bid = 0,tid = 0;
//
//	nfl = (int)((FL/FS*NFFT)+0.5);
//	nfh = (int)((FH/FS*NFFT)+0.5);
//	freqbinnum = *(FreqBin);
//	bid = blockIdx.x;
//	tid = threadIdx.x;
//
//	if(tid==0)
//	{
//		memset(Mabs,0,sizeof(float)*NFFT/2);
//	}
//    // -----------------ÿ���̶߳�ӦƵ��������----------------------
//	//ÿ���̼߳���freqbinnum��Ƶ���������
//	for(int j=0;j<freqbinnum;j++)
//	{
//		tempX=0.0;
//		tempY=0.0;
//		for(int i=0;i<ARNUM;i++)
//		{		
//		   XNSS.x=PhiArray[bid*ARNUM*(NFFT/2)+i*(NFFT/2)+tid*freqbinnum+j].x;
//		   XNSS.y=PhiArray[bid*ARNUM*(NFFT/2)+i*(NFFT/2)+tid*freqbinnum+j].y;
//		   ax=dev_fft[i*(NFFT/2)+(tid*freqbinnum+j)].x;
//		   ay=dev_fft[i*(NFFT/2)+(tid*freqbinnum+j)].y;
//		   bx=XNSS.x;
//		   by=XNSS.y;
//
//		   if (tid*freqbinnum+j>= nfl && tid*freqbinnum+j<=nfh)
//		   {
//				XFFTafterPinYi.x=ax*bx-ay*by;
//				XFFTafterPinYi.y=ax*by+bx*ay;
//		   }
//		   else
//		   {
//				XFFTafterPinYi.x=0;
//				XFFTafterPinYi.y=0;
//		   }
//
//		   tempX=tempX+ XFFTafterPinYi.x; 
//		   tempY=tempY+ XFFTafterPinYi.y;
//		}
//	
//		Mabs[tid*freqbinnum+j]=pow(tempX,2)+pow(tempY,2);
//
//		//�����߳�ͬ��
//		__syncthreads();
//	
//		//-----------------����Ƶ�����������ӵ��������ÿ����ĵ�һ���߳�-----------------	
//		if(tid==0)
//		{
//		   energyEachBoShu=0.0f;
//		   for(int k=0;k<NFFT/2;k++)
//		   {
//			   energyEachBoShu=energyEachBoShu+Mabs[k];
//		   }
//		   dev_energy[bid]= energyEachBoShu;	   
//		}	
//	}
//}

//__global__ void frequency_domain_beamform(hipfftComplex *dev_fft,hipfftReal *dev_energy,hipfftComplex *PhiArray)
//{
//	__shared__ float Mabs[THREADNUMPERBLK];
//	float tempX=0.0f;
//	float tempY=0.0f;
//	int nfl = 0;
//	int nfh = 0;
//	hipComplex XNSS;
//	hipComplex XFFTafterPinYi;
//	float ax = 0.0f,ay=0.0f,bx=0.0f,by=0.0f;
//	float energyEachBoShu = 0.0f;
//	int bid = 0,tid = 0;
//	int beamidx = 0, freqidx = 0;
//
//	nfl = (int)((FL/FS*NFFT)+0.5);
//	nfh = (int)((FH/FS*NFFT)+0.5);
//
//	bid = blockIdx.x;
//	tid = threadIdx.x;
//	beamidx = bid % BeamformNumber;
//	freqidx = bid / BeamformNumber*512+tid;
//
//	if(tid==0)
//	{
//		memset(Mabs,0,sizeof(float)*threadsPerBlock);
//	}
//    // -----------------ÿ���̶߳�ӦƵ��������----------------------
//	tempX=0.0;
//	tempY=0.0;
//	for(int i=0;i<ARNUM;i++)
//	{		
//		XNSS.x=PhiArray[beamidx*ARNUM*(NFFT/2)+i*(NFFT/2)+freqidx].x;
//		XNSS.y=PhiArray[beamidx*ARNUM*(NFFT/2)+i*(NFFT/2)+freqidx].y;
//		ax=dev_fft[i*(NFFT/2+1)+freqidx].x;
//		ay=dev_fft[i*(NFFT/2+1)+freqidx].y;
//		bx=XNSS.x;
//		by=XNSS.y;
//
//		if (freqidx>= nfl && freqidx<=nfh)
//		{
//			XFFTafterPinYi.x=ax*bx-ay*by;
//			XFFTafterPinYi.y=ax*by+bx*ay;
//		}
//		else
//		{
//			XFFTafterPinYi.x=0;
//			XFFTafterPinYi.y=0;
//		}
//
//		tempX=tempX+ XFFTafterPinYi.x; 
//		tempY=tempY+ XFFTafterPinYi.y;
//	}
//	
//	Mabs[tid]=pow(tempX,2)+pow(tempY,2);
//
//	//�����߳�ͬ��
//	__syncthreads();
//	
//	//-----------------����Ƶ�����������ӵ��������ÿ����ĵ�һ���߳�-----------------	
//	if(tid==0)
//	{
//		energyEachBoShu=0.0f;
//		for(int k=0;k<threadsPerBlock;k++)
//		{
//			energyEachBoShu=energyEachBoShu+Mabs[k];
//		}
//		dev_energy[bid]= energyEachBoShu;	   
//		//if(bid == 10+97)
//		//{
//		//	printf("dev_energy[%d] = %.3f\n",bid,dev_energy[bid]);
//		//}
//	}	
//
//}

__global__ void FD_Beamform(hipfftComplex *dev_fft,hipfftReal *dev_energy,hipfftComplex *PhiArray,int nfl,int nfh)
{
	__shared__ float Mabs[THREADNUMPERBLK];
	float      tempX=0.0f;
	float      tempY=0.0f;
	hipComplex  XNSS;
	hipComplex  XFFTafterPinYi;
	float      ax = 0.0f,ay=0.0f,bx=0.0f,by=0.0f;
	float      energyEachBoShu = 0.0f;
	int        bid = 0,tid = 0;
	int        beamidx = 0, freqidx = 0;

	bid = blockIdx.x;
	tid = threadIdx.x;
	beamidx = bid % BEAMNUM;
	freqidx = bid / BEAMNUM*THREADNUMPERBLK+tid;

	if(tid==0)
	{
		memset(Mabs,0,sizeof(float)*THREADNUMPERBLK);
	}
	__syncthreads();

    // -----------------ÿ���̶߳�ӦƵ��������----------------------
	tempX=0.0;
	tempY=0.0;
	for(int i=0;i<ARRAYNUM;i++)
	{		
		XNSS.x=PhiArray[beamidx*ARRAYNUM*(NFFT/2)+i*(NFFT/2)+freqidx].x;
		XNSS.y=PhiArray[beamidx*ARRAYNUM*(NFFT/2)+i*(NFFT/2)+freqidx].y;
		ax=dev_fft[i*(NFFT/2+1)+freqidx].x;
		ay=dev_fft[i*(NFFT/2+1)+freqidx].y;
		bx=XNSS.x;
		by=XNSS.y;

		if (freqidx>= nfl && freqidx<=nfh)
		{
			XFFTafterPinYi.x=ax*bx-ay*by;
			XFFTafterPinYi.y=ax*by+bx*ay;
		}
		else
		{
			XFFTafterPinYi.x=0;
			XFFTafterPinYi.y=0;
		}

		tempX=tempX+ XFFTafterPinYi.x; 
		tempY=tempY+ XFFTafterPinYi.y;
	}

	Mabs[tid]=pow(tempX,2)+pow(tempY,2);

	//�����߳�ͬ��
	__syncthreads();	

	//-----------------����Ƶ�����������ӵ��������ÿ����ĵ�һ���߳�-----------------	
	if(tid==0)
	{
		energyEachBoShu=0.0f;
		for(int k=0;k<THREADNUMPERBLK;k++)
		{
			energyEachBoShu=energyEachBoShu+Mabs[k];
		}
		dev_energy[bid]= energyEachBoShu;	   
	}
}

__global__ void MatrixSumRow(hipfftReal *dev_energy,hipfftReal *sum_energy,int nrow,int ncol)
{
	int bid = 0,tid = 0;
	int row = 0,col = 0;
	float sum = 0.0;
	bid = blockIdx.x;
	row = nrow;
	col = ncol;

	for(int ii = 0;ii<row;ii++)
	{
		sum = sum+dev_energy[ii*col+bid];
	}
	sum_energy[bid] = sum;
}

__global__ void DownSamplingFilter(hipfftComplex *dev_fft_sig,hipfftComplex *dev_fft_filter,hipfftComplex *dev_fft_yk,int FFTN)
{
	int bid = 0,tid = 0;
	hipComplex Sigk;
	hipComplex Hk;
	int chanIdx = 0;
	int freqIdx = 0;

	bid = blockIdx.x;
	tid = threadIdx.x;
	chanIdx = bid % (CHANNUM*2);
	freqIdx = bid / (CHANNUM*2)*THREADNUMPERBLK+tid;
	//for(int ii=0;ii<FFTN/2+1;ii++)
	//{
	//	Sigk.x = dev_fft_sig[bid*FFTN+ii].x;
	//	Sigk.y = dev_fft_sig[bid*FFTN+ii].y;
	//	Hk.x = dev_fft_filter[ii].x;
	//	Hk.y = dev_fft_filter[ii].y;
	//	dev_fft_yk[bid*FFTN+ii].x = Sigk.x*Hk.x-Sigk.y*Hk.y;
	//	dev_fft_yk[bid*FFTN+ii].y = Sigk.x*Hk.y+Sigk.y*Hk.x;
	//}

	Sigk.x = dev_fft_sig[chanIdx*FFTN+freqIdx].x;
	Sigk.y = dev_fft_sig[chanIdx*FFTN+freqIdx].y;
	Hk.x = dev_fft_filter[freqIdx].x;
	Hk.y = dev_fft_filter[freqIdx].y;
	dev_fft_yk[chanIdx*FFTN+freqIdx].x = Sigk.x*Hk.x-Sigk.y*Hk.y;
	dev_fft_yk[chanIdx*FFTN+freqIdx].y = Sigk.x*Hk.y+Sigk.y*Hk.x;

	if( bid/(CHANNUM*2)>= 255 && tid == THREADNUMPERBLK-1)
	{
		Sigk.x = dev_fft_sig[chanIdx*FFTN+FFTN/2].x;
		Sigk.y = dev_fft_sig[chanIdx*FFTN+FFTN/2].y;
		Hk.x = dev_fft_filter[FFTN/2].x;
		Hk.y = dev_fft_filter[FFTN/2].y;
		dev_fft_yk[chanIdx*FFTN+FFTN/2].x = Sigk.x*Hk.x-Sigk.y*Hk.y;
		dev_fft_yk[chanIdx*FFTN+FFTN/2].y = Sigk.x*Hk.y+Sigk.y*Hk.x;
	}
}

__global__ void IFFTNormalize(hipfftReal *dev_fft_yout,hipfftReal *dev_databuff,int FFTN)
{
	int bid = 0,tid = 0;
	int chanIdx = 0;
	int timeIdx = 0;

	bid = blockIdx.x;
	tid = threadIdx.x;

	chanIdx = bid % (CHANNUM*2);
	timeIdx = bid / (CHANNUM*2)*THREADNUMPERBLK+tid+FFTN/4;
	
	//if(bid < CHANNUM*2 && tid == 0)
	//{
	//	memcpy(dev_databuff+chanIdx*FFTN/DOWNSAMPLE,dev_databuff+chanIdx*FFTN/DOWNSAMPLE+FFTN/DOWNSAMPLE/2,FFTN/DOWNSAMPLE/2*sizeof(float));
	//}

	if(timeIdx % DOWNSAMPLE == 0)
	{
		dev_databuff[chanIdx*FFTN/DOWNSAMPLE + FFTN/DOWNSAMPLE/2 + (timeIdx-FFTN/4)/DOWNSAMPLE] = dev_fft_yout[chanIdx*FFTN+timeIdx] / FFTN;
	}
}

__global__ void DelayFilterGen(float *h,int m,float theta,float *tau,int *dI)
{
	int bid = 0,tid = 0;
	int k=0;
	float dfs = 0.0;
	int DI = 0;
	__shared__ float sum;

	bid = blockIdx.x;
	tid = threadIdx.x;

	if(tid == 0)
	{
		sum = 0.0;
		dfs = bid*d*cos(theta/180.0*PI)/UWC*(FS/DOWNSAMPLE);
		DI = int(bid*d*cos(theta/180.0*PI)/UWC*(FS/DOWNSAMPLE)+0.5);
		tau[bid] =dfs-DI;
		dI[bid] = DI;
		//printf("bid=%d,m=%d,theta = %.3f,dfs = %.3f,DI = %d\n",bid,m,theta,dfs,DI);
	}

	//�����߳�ͬ��
	__syncthreads();

	k = tid-m;
	h[bid*(2*m+1)+tid] = sin(k*1.0*PI-tau[bid]*PI+0.000001)/(k*1.0*PI-tau[bid]*PI+0.000001);

	//�����߳�ͬ��
	__syncthreads();

	if(tid == 0)
	{
		for(int k=0;k<2*m+1;k++)
		{
			sum = sum + h[bid*(2*m+1)+k];
		}
	}
	__syncthreads();
	
	h[bid*(2*m+1)+tid] =  h[bid*(2*m+1)+tid]/sum;
}

__global__ void FineDelayFilter(hipfftReal *dev_xin,hipfftReal *dev_yout,hipfftReal *delayfilter,int m)
{
	int bid,tid;
	float x=0.0,h=0.0;
	float sum = 0.0;

	bid = blockIdx.x;
	tid = threadIdx.x;
	__shared__ float y[2*M+1];

	if(tid == 0)
	{
		for(int ii=0;ii<2*m;ii++)
		{
			y[ii] = 0.0;
		}
	}
	
	if(bid-2*m+tid >= 0 && bid-2*m+tid < (FILTER_FRAME/DOWNSAMPLE))
	{
		x = dev_xin[bid-2*m+tid];
	}
	if(2*m-tid >=0)
	{
		h = delayfilter[2*m-tid];
	}
	y[tid] = x*h;

	//if(bid == 24855)
	//{
	//	printf("bid = %d,x=%.8f,h=%.8f,y=%.8f\n",bid,x,h,y);
	//}

	//�����߳�ͬ��
	__syncthreads();
	if(tid == 0)
	{
		sum = 0.0;
		for(int jj=0;jj<2*m+1;jj++)
		{
			sum = sum + y[jj];
		}
		dev_yout[bid] = sum;
		//if(bid == 24855)
		//{
		//	printf("bid = %d,dev_yout=%.8f\n",bid,dev_yout[bid]);
		//}
	}
}

//void Run(LPVOID lParam)
//{
//	LARGE_INTEGER nFreq;
//    LARGE_INTEGER nBeginTime;
//    LARGE_INTEGER nEndTime;
//	FILE *fp=NULL,*fpw=NULL;
//	int nfl = (int)((FL/FS*NFFT)+0.5);
//	int nfh = (int)((FH/FS*NFFT)+0.5);
//	int FreqbinPerThread = (int)((nfh-nfl+1)/512.0 + 0.5);
//	int FrameNum = 416;
//    double time;
//	int BlockRowNum = 0;
//
//	// -------------��ʼ�������������ڴ�---------------------
//	float *a=NULL,*signaldata=NULL;
//	Complex *sk=NULL;
//	float *debugvar = NULL;
//
//	a = (float *)malloc( sizeof(float) * SAMPNUM * ARNUM );
//	memset(a,0, sizeof(float) * SAMPNUM * ARNUM );
//
//	signaldata = (float *)malloc( sizeof(float) * SAMPNUM * ARNUM );
//	memset(signaldata,0, sizeof(float) * SAMPNUM * ARNUM );
//
//	sk = (Complex *)malloc( sizeof(Complex) * (NFFT/2+0) * ARNUM );
//	memset(sk,0,sizeof(Complex) * (NFFT/2+0) * ARNUM);
//
//	debugvar = (float *)malloc( sizeof(float) * ARNUM * BeamformNumber *(NFFT/2+0));
//	memset(debugvar,0, sizeof(float) * ARNUM * BeamformNumber *(NFFT/2+0));
//
//	float c[BeamformNumber]={0.0};
//
//	// --------------��ʼ��������GPU�ڴ�--------------------
//	hipError_t cudaStatus;
//	hipfftReal *dev_a=NULL;
//	hipfftComplex *dev_fft=NULL;
//	hipfftReal *dev_energy=NULL;//�����豸�ϱ���
//	hipfftReal *sum_energy=NULL;//�����豸�ϱ���
//	hipfftComplex *PhiArray = NULL;
//	int MatrixRow= NULL;
//	int MatrixCol= NULL;
//	BlockRowNum = NFFT/2/threadsPerBlock;
//  
//	hipMalloc((void**)&dev_energy,BeamformNumber*BlockRowNum*sizeof(hipfftReal));
//	hipMalloc((void**)&sum_energy,BeamformNumber*sizeof(hipfftReal));
//	hipMalloc((void**)&PhiArray,ARNUM*BeamformNumber*(NFFT/2)*sizeof(hipfftComplex));
//	MatrixRow = BlockRowNum;
//	MatrixCol = BeamformNumber;
//
//	cudaStatus = hipMalloc((void **)&dev_a, sizeof(hipfftReal) * SAMPNUM * ARNUM  );
//	if (cudaStatus != hipSuccess)
//	{
//		printf (" hipMalloc Error! \n ");
//	}
//	cudaStatus = hipMalloc((void **)&dev_fft,  sizeof(hipfftComplex) * (NFFT/2+1) * ARNUM );
//	if (cudaStatus != hipSuccess)
//	{
//		printf (" hipMalloc Error! \n ");
//	}
//
//	hipfftHandle plan;   // �������
//    hipfftPlan1d(&plan, SAMPNUM, HIPFFT_R2C, 1);  // ��һά������и�ֵ
//
//	
//    // ------------ѭ����ʼ------------
//
//	QueryPerformanceFrequency(&nFreq);
//    
//	fp = fopen("D:\\GPUTest\\testdata.bin","rb");
//	fpw = fopen("D:\\GPUTest\\beamenergy.bin","wb");
//
//	PhiShiftFactorGen<<<NFFT/2,BeamformNumber>>>(PhiArray);
//	for(int ii=0;ii<FrameNum;ii++)
//	{
//		fread(a,sizeof(float),SAMPNUM*ARNUM,fp);
//		for(int jj=0;jj<SAMPNUM;jj++)
//		{
//			for(int kk=0;kk<ARNUM;kk++)
//			{
//				signaldata[kk*SAMPNUM+jj] = a[jj*ARNUM+kk];
//			}
//		}
//		hipMemcpy(dev_a, signaldata, sizeof(hipfftReal)*SAMPNUM*ARNUM, hipMemcpyHostToDevice);//ʱ�����ݿ�����GPU	
//		QueryPerformanceCounter(&nBeginTime); 
//		// ------------FFT--------------------
//		for (int ll=0; ll<ARNUM; ll++)		
//		{		
//			hipfftExecR2C(plan, (hipfftReal *)&dev_a[ll*SAMPNUM],(hipfftComplex *)&dev_fft[ll*(NFFT/2+1)]);  //��FFT
//		}
//		
//		frequency_domain_beamform<<<BlockRowNum*BeamformNumber,threadsPerBlock>>>(dev_fft,dev_energy,PhiArray);//�����γ�
//		MatrixSumRow<<<BeamformNumber,1>>>(dev_energy,sum_energy,MatrixRow,MatrixCol);
//
//		////------------�������豸 ������ ������ʾ ------------
//		hipMemcpy(c,sum_energy,BeamformNumber*sizeof(float),hipMemcpyDeviceToHost);
//		QueryPerformanceCounter(&nEndTime);
//		//printf("c[37] = %.3f\n",c[37]);
//
//		fwrite(c,sizeof(float),BeamformNumber,fpw);
//
//		time=(double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart;
//		printf("%f\n",time);
//	}
//	fclose(fp);
//	fclose(fpw);
//	free(a); 
//	free(sk);
//	hipFree(dev_a);
//	hipFree(dev_fft);
//	hipFree(dev_energy);
//	hipDeviceReset();	
//	hipfftDestroy(plan);
//	return;
//}

void ReadBoard1Data(LPVOID lParam)
{
	_Longlong fileindex = 0;
	string FilePath = "D:\\20180201�˲�����������\\20180201\\h\\uwrn\\";
	string FileNamePre = "Board1_ADC_";
	string FileIdx = to_string(fileindex);
	string FileNameSur = ".bin";
	string FileName = FilePath + FileNamePre + FileIdx + FileNameSur;
	int DataFileNum = 18;
	FILE *fp = NULL;
	LARGE_INTEGER nFreq;
    LARGE_INTEGER nBeginTime;
    LARGE_INTEGER nEndTime;
	double dftime = 0.0;
	int readbytes = 0;
	int readbuf[TL*CHANNUM+1];
	int BUF_FLAG=0;
	int *pBuf = NULL;
	int *pCounter = NULL;
	int CounterA = FRAMELEN,CounterB = FRAMELEN;
	int temp = 0;

	QueryPerformanceFrequency(&nFreq);

	if(DataBufA_B1 != NULL)
	{
		free(DataBufA_B1);
		DataBufA_B1 = NULL;
	}
	DataBufA_B1 = (int *)malloc(FRAMELEN*CHANNUM*sizeof(int));
	memset(DataBufA_B1,0,FRAMELEN*CHANNUM*sizeof(int));

	if(DataBufB_B1 != NULL)
	{
		free(DataBufB_B1);
		DataBufB_B1 = NULL;
	}
	DataBufB_B1 = (int *)malloc(FRAMELEN*CHANNUM*sizeof(int));
	memset(DataBufB_B1,0,FRAMELEN*CHANNUM*sizeof(int));

	QueryPerformanceCounter(&nBeginTime); 
	//ÿ�ζ�ȡ1�����ݰ�����17samples*16channels����������Ϊ24bit���ͣ���int�ʹ洢
	for(int ii=0;ii<DataFileNum;ii++)
	{
		fileindex = ii;
		FileIdx = to_string(fileindex);
		FileName = FilePath + FileNamePre + FileIdx + FileNameSur;
		if(fp != NULL)
		{
			fclose(fp);
			fp = NULL;
		}
		fp = fopen(FileName.c_str(),"rb");
		for(int jj=0;jj<8e4;jj++)
		{
			while(dftime < TL*1.0 / FS)
			{
				QueryPerformanceCounter(&nEndTime);
				dftime = (double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart;
			}
			dftime = 0.0;
			nBeginTime = nEndTime;
			fread(readbuf,sizeof(int),TL*CHANNUM+1,fp);
			if(0 == BUF_FLAG)
			{
				pBuf = DataBufA_B1; 
				pCounter = &CounterA;
			}
			else
			{
				pBuf = DataBufB_B1; 
				pCounter = &CounterB;
			}
			if(*(pCounter)>=TL) // TL��������ȫ��д��pBuf
			{
				memcpy(pBuf+FRAMELEN*CHANNUM-(*(pCounter))*CHANNUM,readbuf+1,TL*CHANNUM*sizeof(int));
				*(pCounter) = *(pCounter)-TL;
			}
			else
			{
				temp = TL - *(pCounter);
				//д*(pCounter)��������pBuf
				memcpy(pBuf+FRAMELEN*CHANNUM-(*(pCounter))*CHANNUM,readbuf+1,(*(pCounter))*CHANNUM*sizeof(int));
				//����CounterA��CounterB������
				*(pCounter)= FRAMELEN;
				//дtemp����������һ����
				if(0 == BUF_FLAG) //��ǰΪA����д��B
				{
					memcpy(DataBufB_B1+FRAMELEN*CHANNUM-CounterB*CHANNUM,readbuf+(TL-temp)*CHANNUM+1,temp*CHANNUM*sizeof(int));
					//�޸�B����ֵ
					CounterB = CounterB - temp;
					//�л�����
					BUF_FLAG = 1;
				}
				else //��ǰΪB����д��A
				{
					memcpy(DataBufA_B1+FRAMELEN*CHANNUM-CounterA*CHANNUM,readbuf+(TL-temp)*CHANNUM+1,temp*CHANNUM*sizeof(int));
					//�޸�A����ֵ
					CounterA = CounterA - temp;
					//�л�����
					BUF_FLAG = 0;
				}
				//ʹ�¼���Ч
				SetEvent(g_hReadBoard1ThreadReadyEnvent);
			}
		}
	}
}

void ReadBoard2Data(LPVOID lParam)

{
	_Longlong fileindex = 0;
	string FilePath = "D:\\20180201�˲�����������\\20180201\\h\\uwrn\\";
	string FileNamePre = "Board2_ADC_";
	string FileIdx = to_string(fileindex);
	string FileNameSur = ".bin";
	string FileName = FilePath + FileNamePre + FileIdx + FileNameSur;
	int DataFileNum = 18;
	FILE *fp = NULL;
	LARGE_INTEGER nFreq;
    LARGE_INTEGER nBeginTime;
    LARGE_INTEGER nEndTime;
	double dftime = 0.0;
	int readbytes = 0;
	int readbuf[TL*CHANNUM+1];
	int BUF_FLAG=0;
	int *pBuf = NULL;
	int *pCounter = NULL;
	int CounterA = FRAMELEN,CounterB = FRAMELEN;
	int temp = 0;

	QueryPerformanceFrequency(&nFreq);

	if(DataBufA_B2 != NULL)
	{
		free(DataBufA_B2);
		DataBufA_B2 = NULL;
	}
	DataBufA_B2 = (int *)malloc(FRAMELEN*CHANNUM*sizeof(int));
	memset(DataBufA_B2,0,FRAMELEN*CHANNUM*sizeof(int));

	if(DataBufB_B2 != NULL)
	{
		free(DataBufB_B2);
		DataBufB_B2 = NULL;
	}
	DataBufB_B2 = (int *)malloc(FRAMELEN*CHANNUM*sizeof(int));
	memset(DataBufB_B2,0,FRAMELEN*CHANNUM*sizeof(int));

	//ÿ�ζ�ȡ1�����ݰ�����17samples*16channels����������Ϊ24bit���ͣ���int�ʹ洢
	QueryPerformanceCounter(&nBeginTime); 
	for(int ii=0;ii<DataFileNum;ii++)
	{
		fileindex = ii;
		FileIdx = to_string(fileindex);
		FileName = FilePath + FileNamePre + FileIdx + FileNameSur;
		if(fp != NULL)
		{
			fclose(fp);
			fp = NULL;
		}
		fp = fopen(FileName.c_str(),"rb");
		for(int jj=0;jj<8e4;jj++)
		{
			while(dftime < TL*1.0 / FS)
			{
				QueryPerformanceCounter(&nEndTime);
				dftime = (double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart;
			}
			dftime = 0.0;
			nBeginTime = nEndTime;
			fread(readbuf,sizeof(int),TL*CHANNUM+1,fp);
			if(0 == BUF_FLAG)
			{
				pBuf = DataBufA_B2; 
				pCounter = &CounterA;
			}
			else
			{
				pBuf = DataBufB_B2; 
				pCounter = &CounterB;
			}
			if(*(pCounter)>=TL) // TL��������ȫ��д��pBuf
			{
				memcpy(pBuf+FRAMELEN*CHANNUM-(*(pCounter))*CHANNUM,readbuf+1,TL*CHANNUM*sizeof(int));
				*(pCounter) = *(pCounter)-TL;
			}
			else
			{
				temp = TL - *(pCounter);
				//д*(pCounter)��������pBuf
				memcpy(pBuf+FRAMELEN*CHANNUM-(*(pCounter))*CHANNUM,readbuf+1,(*(pCounter))*CHANNUM*sizeof(int));
				//����CounterA��CounterB������
				*(pCounter)= FRAMELEN;
				//дtemp����������һ����
				if(0 == BUF_FLAG) //��ǰΪA����д��B
				{
					memcpy(DataBufB_B2+FRAMELEN*CHANNUM-CounterB*CHANNUM,readbuf+(TL-temp)*CHANNUM+1,temp*CHANNUM*sizeof(int));
					//�޸�B����ֵ
					CounterB = CounterB - temp;
					//�л�����
					BUF_FLAG = 1;
				}
				else //��ǰΪB����д��A
				{
					memcpy(DataBufA_B2+FRAMELEN*CHANNUM-CounterA*CHANNUM,readbuf+(TL-temp)*CHANNUM+1,temp*CHANNUM*sizeof(int));
					//�޸�A����ֵ
					CounterA = CounterA - temp;
					//�л�����
					BUF_FLAG = 0;
				}
				//ʹ�¼���Ч
				SetEvent(g_hReadBoard2ThreadReadyEnvent);
			}
		}
	}	
}

void DataFormatting(LPVOID lParam)
{
	int retval1 = -1;
	int retval2 = -1;
	int BUF_FLAG = 0;
	int temp = 0;

	if(ChannDataBufA != NULL)
	{
		free(ChannDataBufA);
		ChannDataBufA = NULL;
	}
	ChannDataBufA = (float *)malloc(FRAMELEN*CHANNUM*2*sizeof(float));
	memset(ChannDataBufA,0,FRAMELEN*CHANNUM*2*sizeof(float));

	if(ChannDataBufB != NULL)
	{
		free(ChannDataBufB);
		ChannDataBufB = NULL;
	}
	ChannDataBufB = (float *)malloc(FRAMELEN*CHANNUM*2*sizeof(float));
	memset(ChannDataBufB,0,FRAMELEN*CHANNUM*2*sizeof(float));

	while(1)
	{
		retval1 = WaitForSingleObject(g_hReadBoard1ThreadReadyEnvent,2000);
		retval2 = WaitForSingleObject(g_hReadBoard2ThreadReadyEnvent,2000);
		if(retval1 == WAIT_OBJECT_0 && retval2 == WAIT_OBJECT_0)
		{
			if(BUF_FLAG == 0)
			{
				for(int ii=0;ii<CHANNUM;ii++)
				{
					for(int jj=0;jj<FRAMELEN;jj++)
					{
						temp = DataBufA_B1[jj*CHANNUM+ii];
						temp = temp<<8;
						temp = temp>>8;
						ChannDataBufA[ii*FRAMELEN+jj] = temp*1.0/pow(2.0,23) * 2.5;
						
						temp = DataBufA_B2[jj*CHANNUM+ii];
						temp = temp<<8;
						temp = temp>>8;
						ChannDataBufA[ii*FRAMELEN+jj+FRAMELEN*CHANNUM] = temp*1.0/pow(2.0,23) * 2.5;
					}
				}
				BUF_FLAG = 1;
				SetEvent(g_hFrameDataReadyEnvent);
			}
			else
			{
				for(int ii=0;ii<CHANNUM;ii++)
				{
					for(int jj=0;jj<FRAMELEN;jj++)
					{
						temp = DataBufB_B1[jj*CHANNUM+ii];
						temp = temp<<8;
						temp = temp>>8;
						ChannDataBufB[ii*FRAMELEN+jj] = temp*1.0/pow(2.0,23) * 2.5;
						
						temp = DataBufB_B2[jj*CHANNUM+ii];
						temp = temp<<8;
						temp = temp>>8;
						ChannDataBufB[ii*FRAMELEN+jj+FRAMELEN*CHANNUM] = temp*1.0/pow(2.0,23) * 2.5;
					}
				}
				BUF_FLAG = 0;
				SetEvent(g_hFrameDataReadyEnvent);
			}
		}
		else
		{
			printf("DataRead Timeout!\n");
		}
	}
}

//void ArraySignalProcessing(LPVOID lParam)
//{
//	int retval = -1;
//	int BUF_FLAG = 0;
//	int FrameNum = 0;
//	
//	//-----------------�˲�����������-------------------------------
//	float h[FIRORDER+1] = {0.0};
//	float fl = 100.0f,fh = 10e3f;	
//	hipError_t    cudaStatus;
//	hipfftReal    *dev_x=NULL;              //32ͨ��ԭʼ����
//	hipfftReal    *dev_h=NULL;              //�˲���ϵ��
//	hipfftComplex *dev_fft_x=NULL;          //32ͨ��ԭʼ����FFT
//	hipfftComplex *dev_fft_h=NULL;          //�˲���ϵ��FFT
//	hipfftComplex *dev_fft_y=NULL;          //�˲������FFT
//	hipfftReal    *dev_y=NULL;              //�˲������ԭʼ������ʱ���ź�
//	hipfftReal    *dev_chanbuff=NULL;       //�Դ������ݻ�����
//	float        *FilteredDataout = NULL;
//	float        *DownSamplingData = NULL;
//	hipfftHandle  Hplan;                    //�˲���ϵ��FFT
//	hipfftHandle  Xplan;                    //ͨ��ԭʼ����FFT
//	hipfftHandle  Yplan;                    //�˲���ͨ������FFT
//	//----------------------------------------------------------------
//
//	//--------------------------��ʱ�ӱ���----------------------------
//	LARGE_INTEGER nFreq;
//    LARGE_INTEGER nBeginTime;
//    LARGE_INTEGER nEndTime;
//	double time;
//	hipEvent_t start1;
//	hipEvent_t stop1;
//	float msecTotal = 0.0f;
//	//----------------------------------------------------------------
//
//	//--------------------------Ƶ�����γɲ���----------------------
//	int nfl = (int)((2000.0/(FS/DOWNSAMPLE)*NFFT)+0.5);
//	int nfh = (int)((4000.0/(FS/DOWNSAMPLE)*NFFT)+0.5);
//	int FreqbinPerThread = (int)((nfh-nfl+1)/(THREADNUMPERBLK*1.0) + 0.5);
//	int BlockRowNum = 0;
//	hipfftComplex    *dev_fft=NULL;         //32ͨ���������ź�FFT
//	hipfftReal       *dev_energy=NULL;      //��Ƶ�β���������ÿ��Ƶ��512��Ƶ��
//	hipfftReal       *sum_energy=NULL;      //ȫƵ�β���������Ƶ�����������Ϊ��
//	hipfftComplex    *PhiArray = NULL;      //����Ԫ��Ƶ����������
//	hipfftHandle     Beamplan;              //Ƶ�����γ�FFT
//	float           c[BEAMNUM]={0.0};      //������
//	Complex         *sk=NULL;
//	float           *debugvar = NULL;
//	int             peak[BEAMNUM]={0};
//	int             valley[BEAMNUM]={0};
//	bool            traced[BEAMNUM] = {false};
//	int             tracedbeamIdx = -1;
//	float           pretracedtarget[BEAMNUM] = {0.0};
//	int             pretracedtargetIdx[BEAMNUM] = {-1};
//	int             pretracedtargetNum = 0;
//	int             tracedtargetbeam[MAXTRACETARNUM][2];
//	float           *tracebeam = NULL;
//	int             beammatrix[5][BEAMNUM] = {0};
//	int             i0,i1,i2;
//	float           r0,r1,r2;
//	float           delta_index = 0;  
//	float           tracedtargetangle[3] = {0.0};
//	hipfftReal       *dev_delayFilter = NULL;    //��ͨ��ʱ���˲���ϵ��
//	hipfftReal       *dev_tau = NULL;
//	float           delayfiltercoff[ARRAYNUM*(2*M+1)] = {0.0};
//	float           delaytau[ARRAYNUM] = {0.0};
//	hipfftReal       *dev_delayfilterout = NULL;
//	hipfftReal       *dev_delayfilterbuf = NULL;
//	int             *dev_dI = NULL;
//	int             delaydI[ARRAYNUM] = {0};
//	float           *sourcedata = NULL;
//	float           *shiftdata = NULL;
//	float           *delayfilteroutdata = NULL;
//	hipfftReal       *dev_delaychandata = NULL;
//	hipfftReal       *dev_beamdata = NULL;
//	float           *beamdata = NULL;
//	//----------------------------------------------------------------
//
//	if(DownSamplingDataBufA != NULL)
//	{
//		free(DownSamplingDataBufA);
//		DownSamplingDataBufA = NULL;
//	}
//	DownSamplingDataBufA = (float *)malloc(FILTER_FRAME*CHANNUM*2*sizeof(float));
//	memset(DownSamplingDataBufA,0,FILTER_FRAME*CHANNUM*2*sizeof(float));
//
//	if(DownSamplingDataBufB != NULL)
//	{
//		free(DownSamplingDataBufB);
//		DownSamplingDataBufB = NULL;
//	}
//	DownSamplingDataBufB = (float *)malloc(FILTER_FRAME*CHANNUM*2*sizeof(float));
//	memset(DownSamplingDataBufB,0,FILTER_FRAME*CHANNUM*2*sizeof(float));
//
//
//	//-----------------������-----------------------------------
//	FilteredDataout = (float *)malloc(FILTER_FRAME/DOWNSAMPLE*sizeof(float));
//	memset(FilteredDataout,0,FILTER_FRAME/DOWNSAMPLE*sizeof(float));
//	DownSamplingData = (float *)malloc(FRAMELEN*sizeof(float));
//	memset(DownSamplingData,0,FRAMELEN*sizeof(float));
//
//	Complex *Xk_real = NULL;
//	Xk_real = (Complex *)malloc(FILTER_FRAME*sizeof(Complex));
//	memset(Xk_real,0,FILTER_FRAME*sizeof(Complex));
//
//	FILE *fp = NULL;
//	fp = fopen("BeamEng.bin","wb");
//	FILE *fplog = NULL;
//	fplog = fopen("ProcessLog.txt","w");
//	FILE *fpbeam = NULL;
//	fpbeam = fopen("Beam.bin","wb");
//	int retvalprint = 0;
//
//	//-----------------������-----------------------------------
//	
//    hipfftPlan1d(&Hplan, FILTER_FRAME, HIPFFT_R2C, 1);  
//    hipfftPlan1d(&Xplan, FILTER_FRAME, HIPFFT_R2C, 1);  
//    hipfftPlan1d(&Yplan, FILTER_FRAME, HIPFFT_C2R, 1);  
//
//	cudaStatus = hipMalloc((void **)&dev_x, sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);
//	if (cudaStatus != hipSuccess)
//	{
//		printf (" dev_x hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_x,0,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);
//
//	cudaStatus = hipMalloc((void **)&dev_h, sizeof(hipfftReal)*FILTER_FRAME);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_h hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_h,0,sizeof(hipfftReal)*FILTER_FRAME);
//
//	cudaStatus = hipMalloc((void **)&dev_y, sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_y hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_y,0,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);
//
//	cudaStatus = hipMalloc((void **)&dev_fft_x,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_fft_x hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_fft_x,0,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);
//
//	cudaStatus = hipMalloc((void **)&dev_fft_h,sizeof(hipfftComplex)*FILTER_FRAME);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_fft_h hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_fft_h,0,sizeof(hipfftComplex)*FILTER_FRAME);
//
//	cudaStatus = hipMalloc((void **)&dev_fft_y,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_fft_y hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_fft_y,0,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);
//
//	cudaStatus = hipMalloc((void **)&dev_chanbuff,sizeof(hipfftReal)*FILTER_FRAME/DOWNSAMPLE*CHANNUM*2);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_chanbuff hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_chanbuff,0,sizeof(hipfftReal)*FILTER_FRAME/DOWNSAMPLE*CHANNUM*2);
//
//	fir1(FIRORDER,3,fl,fh,FS,5,h);
//	hipMemcpy(dev_h,h,sizeof(hipfftReal)*FIRORDER,hipMemcpyHostToDevice);
//	hipfftExecR2C(Hplan,(hipfftReal *)&dev_h[0],(hipfftComplex *)&dev_fft_h[0]);
//
//	BlockRowNum = NFFT/2/THREADNUMPERBLK;
//	cudaStatus = hipMalloc((void**)&dev_energy,BEAMNUM*BlockRowNum*sizeof(hipfftReal));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_energy hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_energy,0,BEAMNUM*BlockRowNum*sizeof(hipfftReal));
//
//	cudaStatus = hipMalloc((void**)&sum_energy,BEAMNUM*sizeof(hipfftReal));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("sum_energy hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&sum_energy,0,BEAMNUM*sizeof(hipfftReal));
//
//	cudaStatus = hipMalloc((void**)&PhiArray,ARRAYNUM*BEAMNUM*(NFFT/2)*sizeof(hipfftComplex));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("PhiArray hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&PhiArray,0,ARRAYNUM*BEAMNUM*(NFFT/2)*sizeof(hipfftComplex));
//
//	cudaStatus = hipMalloc((void **)&dev_fft,sizeof(hipfftComplex)*(NFFT/2+1)*ARRAYNUM);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_fft hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_fft,0,sizeof(hipfftComplex)*(NFFT/2+1)*ARRAYNUM);
//
//	hipfftPlan1d(&Beamplan,NFFT,HIPFFT_R2C, 1);
//
//	PhiShiftFactorGen<<<NFFT/2,BEAMNUM>>>(PhiArray);
//
//
//	sk = (Complex *)malloc(sizeof(Complex)*(NFFT/2+1)*ARRAYNUM);
//	memset(sk,0,sizeof(Complex)*(NFFT/2+1)*ARRAYNUM);
//
//	debugvar = (float *)malloc(sizeof(float)*BEAMNUM*BlockRowNum);
//	memset(debugvar,0, sizeof(float)*BEAMNUM*BlockRowNum);
//
//	for(int ii = 0;ii<MAXTRACETARNUM;ii++)
//	{
//		tracedtargetbeam[ii][0] = -1;
//		tracedtargetbeam[ii][1] = -1;
//		tracedtargetangle[ii] = -1.0f;
//	}
//
//	cudaStatus = hipMalloc((void **)&dev_delayFilter,sizeof(hipfftReal)*(2*M+1)*ARRAYNUM);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_delayFilter hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_delayFilter,0,sizeof(hipfftReal)*(2*M+1)*ARRAYNUM);
//
//	cudaStatus = hipMalloc((void **)&dev_tau,sizeof(hipfftReal)*ARRAYNUM);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_tau hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_tau,0,sizeof(hipfftReal)*ARRAYNUM);
//
//	cudaStatus = hipMalloc((void **)&dev_delayfilterout,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE+2*M));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_delayfilterout hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_delayfilterout,0,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE+2*M));
//
//	cudaStatus = hipMalloc((void **)&dev_delayfilterbuf,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_delayfilterbuf hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_delayfilterbuf,0,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE));
//
//	cudaStatus = hipMalloc((void **)&dev_dI,sizeof(int)*ARRAYNUM);
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_dI hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_dI,0,sizeof(int)*ARRAYNUM);
//
//	cudaStatus = hipMalloc((void **)&dev_delaychandata,sizeof(int)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE/2));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_delaychandata hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_delaychandata,0,sizeof(int)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE/2));
//
//	cudaStatus = hipMalloc((void **)&dev_beamdata,sizeof(int)*MAXTRACETARNUM*(FILTER_FRAME/DOWNSAMPLE/2));
//	if (cudaStatus != hipSuccess)
//	{
//		printf ("dev_beamdata hipMalloc Error! \n ");
//	}
//	hipMemset((void **)&dev_beamdata,0,sizeof(int)*MAXTRACETARNUM*(FILTER_FRAME/DOWNSAMPLE/2));
//	
//
//	sourcedata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE)*sizeof(float));
//	memset(sourcedata,0,(FILTER_FRAME/DOWNSAMPLE)*sizeof(float));
//
//	shiftdata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE)*sizeof(float));
//	memset(shiftdata,0,(FILTER_FRAME/DOWNSAMPLE)*sizeof(float));
//
//	delayfilteroutdata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE+2*M)*sizeof(float));
//	memset(delayfilteroutdata,0,(FILTER_FRAME/DOWNSAMPLE+2*M)*sizeof(float));	
//
//	beamdata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float));
//	memset(beamdata,0,(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float));
//
//	QueryPerformanceFrequency(&nFreq);
//	hipEventCreate(&start1);
//	hipEventCreate(&stop1);
//
//	while(1)
//	{
//		retval = WaitForSingleObject(g_hFrameDataReadyEnvent,2000);
//		FrameNum++;
//		
//		if(retval<0)
//		{
//			printf("Timeout!\n");
//			return;
//		}
//
//		//�ƶ�������
//		if(BUF_FLAG == 0)
//		{
//			for(int ii=0;ii<CHANNUM*2;ii++)
//			{
//				memmove(DownSamplingDataBufA+ii*FILTER_FRAME,DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,FRAMELEN*sizeof(float));
//				memcpy(DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,ChannDataBufA+ii*FRAMELEN,FRAMELEN*sizeof(float));
//			}
//			hipMemcpy(dev_x,DownSamplingDataBufA,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2,hipMemcpyHostToDevice);
//			BUF_FLAG = 1;
//		}
//		else
//		{
//			for(int ii=0;ii<CHANNUM*2;ii++)
//			{
//				memmove(DownSamplingDataBufA+ii*FILTER_FRAME,DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,FRAMELEN*sizeof(float));
//				memcpy(DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,ChannDataBufB+ii*FRAMELEN,FRAMELEN*sizeof(float));
//			}
//			hipMemcpy(dev_x,DownSamplingDataBufA,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2,hipMemcpyHostToDevice);
//			BUF_FLAG = 0;
//		}
//		
//		hipEventRecord(start1,NULL);
//
//		//-----------------------------------------(1) �ź��˲�������---------------------------------------------------
//		//4.7ms
//		for(int jj=0;jj<CHANNUM*2;jj++)
//		{
//			hipfftExecR2C(Xplan,(hipfftReal *)&dev_x[jj*FILTER_FRAME],(hipfftComplex *)&dev_fft_x[jj*FILTER_FRAME]);
//		}
//		
//		//Ƶ�����(13ms)
//		DownSamplingFilter<<<CHANNUM*2*(FILTER_FRAME/2/THREADNUMPERBLK),THREADNUMPERBLK>>>(dev_fft_x,dev_fft_h,dev_fft_y,FILTER_FRAME);
//	
//		QueryPerformanceCounter(&nBeginTime); 
//		//���任(105ms)
//		for(int jj=0;jj<CHANNUM*2;jj++)
//		{
//			hipfftExecC2R(Yplan,(hipfftComplex *)&dev_fft_y[jj*FILTER_FRAME],(hipfftReal *)&dev_y[jj*FILTER_FRAME]);
//			hipMemcpy(dev_chanbuff+jj*FILTER_FRAME/DOWNSAMPLE,dev_chanbuff+jj*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2,FILTER_FRAME/DOWNSAMPLE/2*sizeof(float),hipMemcpyDeviceToDevice);
//		}
//		IFFTNormalize<<<CHANNUM*2*(FILTER_FRAME/2/THREADNUMPERBLK),THREADNUMPERBLK>>>(dev_y,dev_chanbuff,FILTER_FRAME);	
//
//		QueryPerformanceCounter(&nEndTime);
//		//-----------------------------------------(1) �ź��˲�����������---------------------------------------------------
//
//
//		//-----------------------------------------(2) Ƶ�����γ�---------------------------------------------------
//
//		//ʹ�û������еĺ�FILTER_FRAME/DOWNSAMPLE/2��������Ƶ�����γɣ����Ʒ�λ
//		for (int ii=0;ii<ARRAYNUM;ii++)		
//		{		
//			hipfftExecR2C(Beamplan,(hipfftReal *)&dev_chanbuff[ii*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2],(hipfftComplex *)&dev_fft[ii*(NFFT/2+1)]);
//		}
//
//		FD_Beamform<<<BlockRowNum*BEAMNUM,THREADNUMPERBLK>>>(dev_fft,dev_energy,PhiArray,nfl,nfh);//�����γ�
//		MatrixSumRow<<<BEAMNUM,1>>>(dev_energy,sum_energy,BlockRowNum,BEAMNUM);
//		
//		hipMemcpy(c,sum_energy,BEAMNUM*sizeof(float),hipMemcpyDeviceToHost);
//		fwrite(c,sizeof(float),BEAMNUM,fp);
//		//-----------------------------------------(2) Ƶ�����γɽ���-----------------------------------------------
//
//
//		//-----------------------------------------(3) �����������------------------------------------------
//		//����������������
//		memset(peak,0,BEAMNUM*sizeof(int));
//		memset(valley,0,BEAMNUM*sizeof(int));
//		findpeak(c,peak,BEAMNUM);
//		findvalley(c,valley,BEAMNUM);
//		bool targetexist = false;
//		//memmove(beammatrix+BEAMNUM,beammatrix,4*BEAMNUM*sizeof(int));
//		memset(pretracedtarget,0,sizeof(float)*BEAMNUM);
//		memset(pretracedtargetIdx,0,sizeof(int)*BEAMNUM);
//		pretracedtargetNum = 0;
//
//		for(int kk=0;kk<BEAMNUM;kk++)
//		{
//			if(peak[kk] == 1)
//			{
//				//�ж��Ƿ��Ѹ��ٸò�������Ŀ��
//				int jj=0;
//				for(jj=0;jj<MAXTRACETARNUM;jj++)
//				{
//					//�����Ƿ��Ѹ���
//					if(abs(tracedtargetbeam[jj][0]-kk)<6 && tracedtargetbeam[jj][0]>0)   //�Ѹ��ٸ�Ŀ�꣬���¸������Ƕ�
//					{
//						break;
//					}
//				}
//				if(jj==MAXTRACETARNUM)  //δ����
//				{
//					targetexist = peakdetection(kk,c,valley,2.0);
//				}
//				else  //�Ѹ��٣����ͼ������
//				{
//					targetexist = peakdetection(kk,c,valley,1.2);
//				}
//				if(targetexist)
//				{
//					pretracedtarget[pretracedtargetNum] = c[kk];
//					pretracedtargetIdx[pretracedtargetNum] = kk;
//					pretracedtargetNum++;
//				}
//			}
//		}
//		rbub(pretracedtarget,pretracedtargetIdx,BEAMNUM);
//
//		if(FrameNum == 115)
//		{
//			FrameNum = FrameNum;
//		}
//		for(int kk=0;kk<pretracedtargetNum;kk++)
//		{
//			int jj=0;
//			for(jj=0;jj<MAXTRACETARNUM;jj++)
//			{
//				//�����Ƿ��Ѹ���
//				if(abs(tracedtargetbeam[jj][0]-pretracedtargetIdx[kk])<6 && tracedtargetbeam[jj][0]>0)   //�Ѹ��ٸ�Ŀ�꣬���¸������Ƕ�
//				{
//					tracedtargetbeam[jj][0] = pretracedtargetIdx[kk];
//					tracedtargetbeam[jj][1] = FrameNum;
//					break;
//				}
//			}
//
//			if(jj==MAXTRACETARNUM)  //δ���ٸ�Ŀ�꣬��һ���յĸ���������
//			{
//				int ii = 0;
//				for(ii=0;ii<MAXTRACETARNUM;ii++)
//				{
//					//�����Ƿ��Ѹ���
//					if(tracedtargetbeam[ii][0] < 0)
//					{
//						break;
//					}
//				}
//				if(ii < MAXTRACETARNUM)           //�п��ø�����
//				{
//					tracedtargetbeam[ii][0] = pretracedtargetIdx[kk];
//					tracedtargetbeam[ii][1] = FrameNum;
//				}
//			}
//		}
//		//������������ն�֡δ���µĸ�����
//		for(int jj=0;jj<MAXTRACETARNUM;jj++)
//		{
//			if(tracedtargetbeam[jj][0] >0 && FrameNum - tracedtargetbeam[jj][1] >= 5)
//			{
//				tracedtargetbeam[jj][0] = -1;
//				tracedtargetbeam[jj][1] = -1;
//				tracedtargetangle[jj] = -1.0f;
//			}
//		}
//		//-----------------------------------------(3) �����������-------------------------------------
//
//
//		//-----------------------------------------(4) �������١����ٲ��� ------------------------------
//		for(int jj = 0;jj<MAXTRACETARNUM;jj++)
//		{
//			if(tracedtargetbeam[jj][0] >0)   //�и���Ŀ��
//			{
//				//�����ڲ�
//				i0 = tracedtargetbeam[jj][0]-1;
//				i1 = tracedtargetbeam[jj][0];
//				i2 = tracedtargetbeam[jj][0]+1;
//				r0 = c[i0];
//				r1 = c[i1];
//				r2 = c[i2];
//				delta_index = (r2-r0)/(4*r1-2*r0-2*r2);
//				tracedtargetangle[jj] = (i1+delta_index)*180.0/BEAMNUM;
//				DelayFilterGen<<<ARRAYNUM,2*M+1>>>(dev_delayFilter,M,tracedtargetangle[jj],dev_tau,dev_dI);
//				//DelayFilterGen<<<ARRAYNUM,2*M+1>>>(dev_delayFilter,M,60.292690,dev_tau,dev_dI);
//				hipMemcpy(delayfiltercoff,dev_delayFilter,sizeof(hipfftReal)*ARRAYNUM*(2*M+1),hipMemcpyDeviceToHost);
//				hipMemcpy(delaytau,dev_tau,sizeof(hipfftReal)*ARRAYNUM,hipMemcpyDeviceToHost);
//				hipMemcpy(delaydI,dev_dI,sizeof(int)*ARRAYNUM,hipMemcpyDeviceToHost);
//				
//				for(int kk = 0;kk<ARRAYNUM;kk++)
//				{
//					if(delaydI[kk] >= 0)
//					{
//						hipMemcpy(dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE)+delaydI[kk],dev_chanbuff+kk*(FILTER_FRAME/DOWNSAMPLE),sizeof(hipfftReal)*((FILTER_FRAME/DOWNSAMPLE)-delaydI[kk]),hipMemcpyDeviceToDevice);
//					}
//					else
//					{
//						hipMemcpy(dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE),dev_chanbuff+kk*(FILTER_FRAME/DOWNSAMPLE)-delaydI[kk],sizeof(hipfftReal)*((FILTER_FRAME/DOWNSAMPLE)+delaydI[kk]),hipMemcpyDeviceToDevice);
//					}
//
//					//hipMemcpy(sourcedata,dev_chanbuff+kk*(FILTER_FRAME/DOWNSAMPLE),(FILTER_FRAME/DOWNSAMPLE)*sizeof(float),hipMemcpyDeviceToHost);
//					//hipMemcpy(shiftdata,dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE),(FILTER_FRAME/DOWNSAMPLE)*sizeof(float),hipMemcpyDeviceToHost);
//
//					if(fabs(delaytau[kk]) > 0.0001)
//					{
//						FineDelayFilter<<<(FILTER_FRAME/DOWNSAMPLE+2*M),2*M+1>>>((hipfftReal *)&dev_delayfilterbuf[kk*FILTER_FRAME/DOWNSAMPLE],(hipfftReal *)&dev_delayfilterout[kk*(FILTER_FRAME/DOWNSAMPLE+2*M)],(hipfftReal *)&dev_delayFilter[kk*(2*M+1)],M);
//					}
//					else
//					{
//						hipMemcpy(dev_delayfilterout+kk*(FILTER_FRAME/DOWNSAMPLE+2*M)+M,dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE),sizeof(hipfftReal)*(FILTER_FRAME/DOWNSAMPLE),hipMemcpyDeviceToDevice);
//					}
//					hipMemcpy(dev_delaychandata+kk*(FILTER_FRAME/DOWNSAMPLE/2),dev_delayfilterout+kk*(FILTER_FRAME/DOWNSAMPLE+2*M)+M+FILTER_FRAME/DOWNSAMPLE/4,sizeof(hipfftReal)*FILTER_FRAME/DOWNSAMPLE/2,hipMemcpyDeviceToDevice);
//					//hipMemcpy(delayfilteroutdata,dev_delayfilterout+kk*(FILTER_FRAME/DOWNSAMPLE+2*M),(FILTER_FRAME/DOWNSAMPLE+M*2)*sizeof(float),hipMemcpyDeviceToHost);					
//					//if(FrameNum==2)
//					//{
//					//	FrameNum = FrameNum;
//					//}
//				}
//				MatrixSumRow<<<FILTER_FRAME/DOWNSAMPLE/2,1>>>(dev_delaychandata,dev_beamdata+jj*FILTER_FRAME/DOWNSAMPLE/2,ARRAYNUM,FILTER_FRAME/DOWNSAMPLE/2);
//				hipMemcpy(beamdata,dev_beamdata+jj*FILTER_FRAME/DOWNSAMPLE/2,FILTER_FRAME/DOWNSAMPLE/2*sizeof(float),hipMemcpyDeviceToHost);
//				fwrite(beamdata,sizeof(float),FILTER_FRAME/DOWNSAMPLE/2,fpbeam);
//			}
//		}
//
//		hipEventRecord(stop1,NULL);
//		hipEventSynchronize(stop1);
//		//time=(double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart;
//		hipEventElapsedTime(&msecTotal,start1,stop1);
//		printf("%d:%f;%d,%d;%d,%d;%d,%d\n",FrameNum,msecTotal,tracedtargetbeam[0][0],tracedtargetbeam[0][1],tracedtargetbeam[1][0],tracedtargetbeam[1][1],tracedtargetbeam[2][0],tracedtargetbeam[2][1]);
//		fprintf(fplog,"%d:%f;%d,%d;%d,%d;%d,%d\n",FrameNum,msecTotal,tracedtargetbeam[0][0],tracedtargetbeam[0][1],tracedtargetbeam[1][0],tracedtargetbeam[1][1],tracedtargetbeam[2][0],tracedtargetbeam[2][1]);
//		fflush(fplog);
//	}
//	fclose(fp);
//	fp = NULL;
//	fclose(fplog);
//	fplog = NULL;
//	fclose(fpbeam);
//	fpbeam = NULL;
//}
__global__ void Psd(hipfftComplex *Xk,hipfftReal *Xabs, int N)
{
    int bid = 0,tid = 0;
    int freqIdx = 0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    freqIdx = bid*THREADNUMPERBLK+tid;

    Xabs[freqIdx] = (Xk[freqIdx].x*Xk[freqIdx].x+Xk[freqIdx].y*Xk[freqIdx].y) / N;
}

__global__ void PsdAverage(hipfftReal *Xabs,hipfftReal *Xk_avg)
{
    int bid = 0,tid = 0;
    int freqIdx = 0;
	float sum = 0.0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    freqIdx = bid*THREADNUMPERBLK+tid;

	for(int ii = 0;ii<PSD_AVG_NUM;ii++)
	{
		sum += Xabs[ii*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)+freqIdx] / PSD_AVG_NUM;
	}
	Xk_avg[freqIdx] = 10*log10((sum+EPS)/1e-12);
}

__global__ void PsdSub(hipfftReal *Xk_avg,hipfftReal *Xk_smooth,hipfftReal *Xk_diff,int idx1,int idx2)
{
    int bid = 0,tid = 0;
    int freqIdx = 0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    freqIdx = bid*THREADNUMPERBLK+tid;
	if(freqIdx >= idx1 && freqIdx <= idx2)
	{
		Xk_diff[freqIdx] = Xk_avg[freqIdx] - Xk_smooth[freqIdx];
	}
	else
	{
		Xk_diff[freqIdx] = 0;
	}
	//if(freqIdx == 50000)
	//{
	//	printf("Xk_smooth=%.5f\n",Xk_smooth[freqIdx]);
	//}
}
//__global__ void PsdLog(hipfftReal *Xk_avg)
//{
//    int bid = 0,tid = 0;
//    int freqIdx = 0;
//	float sum = 0.0;
//
//    bid = blockIdx.x;
//    tid = threadIdx.x;
//
//    freqIdx = bid*THREADNUMPERBLK+tid;
//
//	Xk_avg[freqIdx] = 10*log10(Xk_avg[freqIdx]+EPS);
//}

__global__ void FrequencyDomainFilter(hipfftComplex *Xk,float deltaf,float StartFreq,float EndFreq)
{
    int bid = 0,tid = 0;
    int freqIdx = 0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    freqIdx = bid*THREADNUMPERBLK+tid;
	if(freqIdx * deltaf < StartFreq || freqIdx * deltaf > EndFreq)
	{
		Xk[freqIdx].x = 0;
		Xk[freqIdx].y = 0;
	}
	//else
	//{
	//	printf("Xk[freqIdx].x = %.6f\n",Xk[freqIdx].x);
	//}
}

__global__ void SignalSqr(hipfftReal *X)
{
    int bid = 0,tid = 0;
    int sigIdx = 0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    sigIdx = bid*THREADNUMPERBLK+tid;
	X[sigIdx] = X[sigIdx]*X[sigIdx];
}

__global__ void DemonAdd(hipfftComplex *Xk,hipfftReal *Xabs, int N)
{
    int bid = 0,tid = 0;
    int freqIdx = 0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    freqIdx = bid*THREADNUMPERBLK+tid;

    Xabs[freqIdx] += (Xk[freqIdx].x*Xk[freqIdx].x+Xk[freqIdx].y*Xk[freqIdx].y) / N;
}

__global__ void DemonSub(hipfftReal *Xk_avg,hipfftReal *Xk_smooth,hipfftReal *Xk_diff)
{
    int bid = 0,tid = 0;
    int freqIdx = 0;

    bid = blockIdx.x;
    tid = threadIdx.x;

    freqIdx = bid;
	Xk_diff[freqIdx] = Xk_avg[freqIdx] - Xk_smooth[freqIdx];
	if(Xk_diff[freqIdx] < 0)
	{
		Xk_diff[freqIdx] = 0;
	}
}

float VectorThetSPF(hipfftComplex P_f, hipfftComplex Vx_f, hipfftComplex Vy_f)
{
	float fTheta=0.0;
	float sina=-P_f.y*Vy_f.x+P_f.x*Vy_f.y;
	float cosa=-P_f.y*Vx_f.x+P_f.x*Vx_f.y;
	fTheta=atan2(sina, cosa)*180/PI;
	return fTheta;
}

void ArraySignalProcessing(LPVOID lParam)
{
	int retval = -1;
	int BUF_FLAG = 0;
	int FrameNum = 0;
	
	//-----------------�˲�����������-------------------------------
	float h[FIRORDER+1] = {0.0};
	float fl = 100.0f,fh = 10e3f;	
	hipError_t    cudaStatus;
	hipfftReal    *dev_x=NULL;              //32ͨ��ԭʼ����
	hipfftReal    *dev_h=NULL;              //�˲���ϵ��
	hipfftComplex *dev_fft_x=NULL;          //32ͨ��ԭʼ����FFT
	hipfftComplex *dev_fft_h=NULL;          //�˲���ϵ��FFT
	hipfftComplex *dev_fft_y=NULL;          //�˲������FFT
	hipfftReal    *dev_y=NULL;              //�˲������ԭʼ������ʱ���ź�
	hipfftReal    *dev_chanbuff=NULL;       //�Դ������ݻ�����
	float        *FilteredDataout = NULL;
	float        *DownSamplingData = NULL;
	hipfftHandle  Hplan;                    //�˲���ϵ��FFT
	hipfftHandle  Xplan;                    //ͨ��ԭʼ����FFT
	hipfftHandle  Yplan;                    //�˲���ͨ������FFT
	//----------------------------------------------------------------

	//--------------------------��ʱ�ӱ���----------------------------
	LARGE_INTEGER nFreq;
    LARGE_INTEGER nBeginTime;
    LARGE_INTEGER nEndTime;
	double time;
	hipEvent_t start1;
	hipEvent_t stop1;
	float msecTotal = 0.0f;
	//----------------------------------------------------------------

	//--------------------------Ƶ�����γɲ���----------------------
	int nfl = (int)((2000.0/(FS/DOWNSAMPLE)*NFFT)+0.5);
	int nfh = (int)((4000.0/(FS/DOWNSAMPLE)*NFFT)+0.5);
	int FreqbinPerThread = (int)((nfh-nfl+1)/(THREADNUMPERBLK*1.0) + 0.5);
	int BlockRowNum = 0;
	hipfftComplex    *dev_fft=NULL;         //32ͨ���������ź�FFT
	hipfftReal       *dev_energy=NULL;      //��Ƶ�β���������ÿ��Ƶ��512��Ƶ��
	hipfftReal       *sum_energy=NULL;      //ȫƵ�β���������Ƶ�����������Ϊ��
	hipfftComplex    *PhiArray = NULL;      //����Ԫ��Ƶ����������
	hipfftHandle     Beamplan;              //Ƶ�����γ�FFT
	float           c[BEAMNUM]={0.0};      //������
	Complex         *sk=NULL;
	float           *debugvar = NULL;
	int             peak[BEAMNUM]={0};
	int             valley[BEAMNUM]={0};
	bool            traced[BEAMNUM] = {false};
	int             tracedbeamIdx = -1;
	float           pretracedtarget[BEAMNUM] = {0.0};
	int             pretracedtargetIdx[BEAMNUM] = {-1};
	int             pretracedtargetNum = 0;
	int             tracedtargetbeam[MAXTRACETARNUM][2];
	float           *tracebeam = NULL;
	int             beammatrix[5][BEAMNUM] = {0};
	int             i0,i1,i2;
	float           r0,r1,r2;
	float           delta_index = 0;  
	float           tracedtargetangle[3] = {0.0};
	hipfftReal       *dev_delayFilter = NULL;    //��ͨ��ʱ���˲���ϵ��
	hipfftReal       *dev_tau = NULL;
	float           delayfiltercoff[ARRAYNUM*(2*M+1)] = {0.0};
	float           delaytau[ARRAYNUM] = {0.0};
	hipfftReal       *dev_delayfilterout = NULL;
	hipfftReal       *dev_delayfilterbuf = NULL;
	int             *dev_dI = NULL;
	int             delaydI[ARRAYNUM] = {0};
	float           *sourcedata = NULL;
	float           *shiftdata = NULL;
	float           *delayfilteroutdata = NULL;
	hipfftReal       *dev_delaychandata = NULL;
	hipfftReal       *dev_beamdata = NULL;
	float           *beamdata = NULL;
	//----------------------------------------------------------------
    //----------------------------Psd and DEMON-----------------------
    hipfftReal       *dev_tracebeam=NULL;
    hipfftComplex    *dev_tracebeam_spec=NULL;
    hipfftReal       *dev_tracebeam_psd=NULL;
	hipfftReal       *dev_tracebeam_psd_avg = NULL;
    hipfftComplex    *dev_tracebeam_demonspec=NULL;
	hipfftComplex    *dev_tracebeam_demonspec_band=NULL;
    hipfftReal       *dev_tracebeam_demon=NULL;
    hipfftReal       *dev_tracebeam_demon_band_data=NULL;
    hipfftHandle     PSDplan;
    hipfftHandle     DEMONplan;
	hipfftHandle     DEMONBandplan;
    float           *trace_beam_psd = NULL;
	float           fDf;
	int             idx1;
	int             idx2;
	int             idxLen;
    float           *trace_beam_psd_smooth = NULL;
	hipfftReal       *dev_tracebeam_psd_S = NULL;
	hipfftReal       *dev_tracebeam_psd_E = NULL;
	float           fPsdEVar=0.0;
    float           *trace_beam_demon = NULL;
	float           *trace_beam_demon_smooth = NULL;
	hipfftReal       *dev_trace_beam_demon_cut = NULL;
	hipfftReal       *dev_tracebeam_demon_S = NULL;
	hipfftReal       *dev_tracebeam_demon_E = NULL;
	float           fDemonEVar=0.0;
	//-----------------------ʸ��ͨ������-----------------------------
    hipfftReal       *dev_vector_p_buf=NULL;
    hipfftReal       *dev_vector_x_buf=NULL;
    hipfftReal       *dev_vector_y_buf=NULL;
    hipfftComplex    *dev_vector_p_spec=NULL;
    hipfftComplex    *dev_vector_x_spec=NULL;
    hipfftComplex    *dev_vector_y_spec=NULL;
    hipfftReal       *dev_vector_p_psd =NULL;
	hipfftReal       *dev_vector_psd_avg=NULL;
    float           *vector_p_psd = NULL;
    float           *vector_p_psd_smooth = NULL;
	hipfftReal       *dev_vector_p_psd_S = NULL;
	hipfftReal       *dev_vector_p_psd_E = NULL;
	float           fVectorPsdEVar=0.0;
    //----------------------------------------------------------------
	if(DownSamplingDataBufA != NULL)
	{
		free(DownSamplingDataBufA);
		DownSamplingDataBufA = NULL;
	}
	DownSamplingDataBufA = (float *)malloc(FILTER_FRAME*CHANNUM*2*sizeof(float));
	memset(DownSamplingDataBufA,0,FILTER_FRAME*CHANNUM*2*sizeof(float));

	if(DownSamplingDataBufB != NULL)
	{
		free(DownSamplingDataBufB);
		DownSamplingDataBufB = NULL;
	}
	DownSamplingDataBufB = (float *)malloc(FILTER_FRAME*CHANNUM*2*sizeof(float));
	memset(DownSamplingDataBufB,0,FILTER_FRAME*CHANNUM*2*sizeof(float));


	//-----------------������-----------------------------------
	FilteredDataout = (float *)malloc(FILTER_FRAME/DOWNSAMPLE*sizeof(float));
	memset(FilteredDataout,0,FILTER_FRAME/DOWNSAMPLE*sizeof(float));
	DownSamplingData = (float *)malloc(FRAMELEN*sizeof(float));
	memset(DownSamplingData,0,FRAMELEN*sizeof(float));

	hipfftComplex *Xk_real = NULL;
	Xk_real = (hipfftComplex *)malloc(FILTER_FRAME*sizeof(hipfftComplex));
	memset(Xk_real,0,FILTER_FRAME*sizeof(hipfftComplex));

	FILE *fp = NULL;
	fp = fopen("BeamEng.bin","wb");
	FILE *fplog = NULL;
	fplog = fopen("ProcessLog.txt","w");
	FILE *fpbeam = NULL;
	fpbeam = fopen("Beam.bin","wb");
	int retvalprint = 0;

	//-----------------������-----------------------------------
	
    hipfftPlan1d(&Hplan, FILTER_FRAME, HIPFFT_R2C, 1);  
    hipfftPlan1d(&Xplan, FILTER_FRAME, HIPFFT_R2C, 1);  
    hipfftPlan1d(&Yplan, FILTER_FRAME, HIPFFT_C2R, 1);  
    hipfftPlan1d(&PSDplan,   PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2), HIPFFT_R2C, 1);
    hipfftPlan1d(&DEMONplan, PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2), HIPFFT_R2C, 1);
    hipfftPlan1d(&DEMONBandplan, PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2), HIPFFT_C2R, 1);  

	cudaStatus = hipMalloc((void **)&dev_x, sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);
	if (cudaStatus != hipSuccess)
	{
		printf (" dev_x hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_x,0,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);

	cudaStatus = hipMalloc((void **)&dev_h, sizeof(hipfftReal)*FILTER_FRAME);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_h hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_h,0,sizeof(hipfftReal)*FILTER_FRAME);

	cudaStatus = hipMalloc((void **)&dev_y, sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_y hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_y,0,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2);

	cudaStatus = hipMalloc((void **)&dev_fft_x,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_fft_x hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_fft_x,0,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);

	cudaStatus = hipMalloc((void **)&dev_fft_h,sizeof(hipfftComplex)*FILTER_FRAME);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_fft_h hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_fft_h,0,sizeof(hipfftComplex)*FILTER_FRAME);

	cudaStatus = hipMalloc((void **)&dev_fft_y,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_fft_y hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_fft_y,0,sizeof(hipfftComplex)*FILTER_FRAME*CHANNUM*2);

	cudaStatus = hipMalloc((void **)&dev_chanbuff,sizeof(hipfftReal)*FILTER_FRAME/DOWNSAMPLE*CHANNUM*2);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_chanbuff hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_chanbuff,0,sizeof(hipfftReal)*FILTER_FRAME/DOWNSAMPLE*CHANNUM*2);

	fir1(FIRORDER,3,fl,fh,FS,5,h);
	hipMemcpy(dev_h,h,sizeof(hipfftReal)*FIRORDER,hipMemcpyHostToDevice);
	hipfftExecR2C(Hplan,(hipfftReal *)&dev_h[0],(hipfftComplex *)&dev_fft_h[0]);

	BlockRowNum = NFFT/2/THREADNUMPERBLK;
	cudaStatus = hipMalloc((void**)&dev_energy,BEAMNUM*BlockRowNum*sizeof(hipfftReal));
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_energy hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_energy,0,BEAMNUM*BlockRowNum*sizeof(hipfftReal));

	cudaStatus = hipMalloc((void**)&sum_energy,BEAMNUM*sizeof(hipfftReal));
	if (cudaStatus != hipSuccess)
	{
		printf ("sum_energy hipMalloc Error! \n ");
	}
	hipMemset((void **)&sum_energy,0,BEAMNUM*sizeof(hipfftReal));

	cudaStatus = hipMalloc((void**)&PhiArray,ARRAYNUM*BEAMNUM*(NFFT/2)*sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		printf ("PhiArray hipMalloc Error! \n ");
	}
	hipMemset((void **)&PhiArray,0,ARRAYNUM*BEAMNUM*(NFFT/2)*sizeof(hipfftComplex));

	cudaStatus = hipMalloc((void **)&dev_fft,sizeof(hipfftComplex)*(NFFT/2+1)*ARRAYNUM);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_fft hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_fft,0,sizeof(hipfftComplex)*(NFFT/2+1)*ARRAYNUM);

	hipfftPlan1d(&Beamplan,NFFT,HIPFFT_R2C, 1);

	PhiShiftFactorGen<<<NFFT/2,BEAMNUM>>>(PhiArray);


    sk = (hipfftComplex *)malloc(sizeof(hipfftComplex)*(NFFT/2+1)*ARRAYNUM);
    memset(sk,0,sizeof(hipfftComplex)*(NFFT/2+1)*ARRAYNUM);

	debugvar = (float *)malloc(sizeof(float)*BEAMNUM*BlockRowNum);
	memset(debugvar,0, sizeof(float)*BEAMNUM*BlockRowNum);

	for(int ii = 0;ii<MAXTRACETARNUM;ii++)
	{
		tracedtargetbeam[ii][0] = -1;
		tracedtargetbeam[ii][1] = -1;
		tracedtargetangle[ii] = -1.0f;
	}

	cudaStatus = hipMalloc((void **)&dev_delayFilter,sizeof(hipfftReal)*(2*M+1)*ARRAYNUM);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_delayFilter hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_delayFilter,0,sizeof(hipfftReal)*(2*M+1)*ARRAYNUM);

	cudaStatus = hipMalloc((void **)&dev_tau,sizeof(hipfftReal)*ARRAYNUM);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_tau hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_tau,0,sizeof(hipfftReal)*ARRAYNUM);

	cudaStatus = hipMalloc((void **)&dev_delayfilterout,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE+2*M));
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_delayfilterout hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_delayfilterout,0,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE+2*M));

	cudaStatus = hipMalloc((void **)&dev_delayfilterbuf,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE));
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_delayfilterbuf hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_delayfilterbuf,0,sizeof(hipfftReal)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE));

	cudaStatus = hipMalloc((void **)&dev_dI,sizeof(int)*ARRAYNUM);
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_dI hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_dI,0,sizeof(int)*ARRAYNUM);

	cudaStatus = hipMalloc((void **)&dev_delaychandata,sizeof(int)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE/2));
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_delaychandata hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_delaychandata,0,sizeof(int)*ARRAYNUM*(FILTER_FRAME/DOWNSAMPLE/2));

	cudaStatus = hipMalloc((void **)&dev_beamdata,sizeof(int)*MAXTRACETARNUM*(FILTER_FRAME/DOWNSAMPLE/2));
	if (cudaStatus != hipSuccess)
	{
		printf ("dev_beamdata hipMalloc Error! \n ");
	}
	hipMemset((void **)&dev_beamdata,0,sizeof(int)*MAXTRACETARNUM*(FILTER_FRAME/DOWNSAMPLE/2));
	

	sourcedata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE)*sizeof(float));
	memset(sourcedata,0,(FILTER_FRAME/DOWNSAMPLE)*sizeof(float));

	shiftdata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE)*sizeof(float));
	memset(shiftdata,0,(FILTER_FRAME/DOWNSAMPLE)*sizeof(float));

	delayfilteroutdata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE+2*M)*sizeof(float));
	memset(delayfilteroutdata,0,(FILTER_FRAME/DOWNSAMPLE+2*M)*sizeof(float));	

	beamdata = (float *)malloc((FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float));
	memset(beamdata,0,(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float));

    cudaStatus = hipMalloc((void **)&dev_tracebeam,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);

    cudaStatus = hipMalloc((void **)&dev_tracebeam_spec,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_spec hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_spec,0,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);

    cudaStatus = hipMalloc((void **)&dev_tracebeam_psd,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM*PSD_AVG_NUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_psd hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_psd,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM*PSD_AVG_NUM);

	cudaStatus = hipMalloc((void **)&dev_tracebeam_psd_avg,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_psd_avg hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_psd_avg,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);

    cudaStatus = hipMalloc((void **)&dev_tracebeam_demonspec,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_demonspec hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_demonspec,0,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);

    cudaStatus = hipMalloc((void **)&dev_tracebeam_demonspec_band,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_demonspec_band hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_demonspec_band,0,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

    cudaStatus = hipMalloc((void **)&dev_tracebeam_demon,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_demon hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_demon,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);

    cudaStatus = hipMalloc((void **)&dev_tracebeam_demon_band_data,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_demon_band_data hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_demon_band_data,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

    trace_beam_psd = (float *)malloc(PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));
    memset(trace_beam_psd,0,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));

	trace_beam_psd_smooth = (float *)malloc(PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));
    memset(trace_beam_psd_smooth,0,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));

	cudaStatus = hipMalloc((void **)&dev_tracebeam_psd_S,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_psd_S hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_psd_S,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);

	cudaStatus = hipMalloc((void **)&dev_tracebeam_psd_E,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_psd_E hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_psd_E,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);	

	trace_beam_demon = (float *)malloc(PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float));
    memset(trace_beam_demon,0,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float));

	trace_beam_demon_smooth = (float *)malloc(DEM_RST_LEN*sizeof(float));
    memset(trace_beam_demon_smooth,0,DEM_RST_LEN*sizeof(float));

	cudaStatus = hipMalloc((void **)&dev_tracebeam_demon_S,sizeof(hipfftReal)*DEM_RST_LEN);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_demon_S hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_demon_S,0,sizeof(hipfftReal)*DEM_RST_LEN);	

	cudaStatus = hipMalloc((void **)&dev_tracebeam_demon_E,sizeof(hipfftReal)*DEM_RST_LEN);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_tracebeam_demon_E hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_tracebeam_demon_E,0,sizeof(hipfftReal)*DEM_RST_LEN);	

	cudaStatus = hipMalloc((void **)&dev_trace_beam_demon_cut,sizeof(hipfftReal)*DEM_RST_LEN);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_trace_beam_demon_cut hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_trace_beam_demon_cut,0,sizeof(hipfftReal)*DEM_RST_LEN);	

	//------------------------------ʸ��ͨ������------------------------------------------------
	cudaStatus = hipMalloc((void **)&dev_vector_p_buf,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_p_buf hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_p_buf,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

	cudaStatus = hipMalloc((void **)&dev_vector_x_buf,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_x_buf hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_x_buf,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

	cudaStatus = hipMalloc((void **)&dev_vector_y_buf,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_y_buf hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_y_buf,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

    cudaStatus = hipMalloc((void **)&dev_vector_p_psd,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*PSD_AVG_NUM);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_p_psd hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_p_psd,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*PSD_AVG_NUM);

    cudaStatus = hipMalloc((void **)&dev_vector_p_spec,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_p_spec hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_p_spec,0,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

    cudaStatus = hipMalloc((void **)&dev_vector_x_spec,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_x_spec hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_x_spec,0,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

    cudaStatus = hipMalloc((void **)&dev_vector_y_spec,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_y_spec hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_y_spec,0,sizeof(hipfftComplex)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));

    cudaStatus = hipMalloc((void **)&dev_vector_psd_avg,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_psd_avg hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_psd_avg,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));	

	vector_p_psd = (float*)malloc(PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));
	memset(vector_p_psd,0,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));

	vector_p_psd_smooth = (float*)malloc(PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));
	memset(vector_p_psd_smooth,0,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float));

	cudaStatus = hipMalloc((void **)&dev_vector_p_psd_S,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_p_psd_S hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_p_psd_S,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);

	cudaStatus = hipMalloc((void **)&dev_vector_p_psd_E,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);
    if (cudaStatus != hipSuccess)
    {
        printf ("dev_vector_p_psd_E hipMalloc Error! \n ");
    }
    hipMemset((void **)&dev_vector_p_psd_E,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);	

	//--------------------------------------------------------------------------------------------
	fDf=FS/DOWNSAMPLE*1.0/(PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
	idx1=(int)(10/fDf);
	idx2=(int)(5000/fDf);
	idxLen=idx2-idx1+1;

    DemFreqBandNum  = 4;
    DemStartFreq[0] = 2000.0; 
    DemEndFreq[0]   = 4000.0;

    DemStartFreq[1] = 4000.0; 
    DemEndFreq[1]   = 6000.0;

	DemStartFreq[2] = 6000.0; 
    DemEndFreq[2]   = 8000.0;

    DemStartFreq[3] = 8000.0; 
    DemEndFreq[3]   = 10000.0;
    //���ɻ�������������ʽ���ʱ���ù淶��������
	CalSmoothPara(&fSmoothA[0][0]);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);

	while(1)
	{
		retval = WaitForSingleObject(g_hFrameDataReadyEnvent,2000);
		FrameNum++;
		
		if(retval<0)
		{
			printf("Timeout!\n");
			return;
		}

		//�ƶ�������
		if(BUF_FLAG == 0)
		{
			for(int ii=0;ii<CHANNUM*2;ii++)
			{
				memmove(DownSamplingDataBufA+ii*FILTER_FRAME,DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,FRAMELEN*sizeof(float));
				memcpy(DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,ChannDataBufA+ii*FRAMELEN,FRAMELEN*sizeof(float));
			}
			hipMemcpy(dev_x,DownSamplingDataBufA,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2,hipMemcpyHostToDevice);
			BUF_FLAG = 1;
		}
		else
		{
			for(int ii=0;ii<CHANNUM*2;ii++)
			{
				memmove(DownSamplingDataBufA+ii*FILTER_FRAME,DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,FRAMELEN*sizeof(float));
				memcpy(DownSamplingDataBufA+ii*FILTER_FRAME+FRAMELEN,ChannDataBufB+ii*FRAMELEN,FRAMELEN*sizeof(float));
			}
			hipMemcpy(dev_x,DownSamplingDataBufA,sizeof(hipfftReal)*FILTER_FRAME*CHANNUM*2,hipMemcpyHostToDevice);
			BUF_FLAG = 0;
		}
		
		hipEventRecord(start1,NULL);

		//-----------------------------------------(1) �ź��˲�������---------------------------------------------------
		//4.7ms
		for(int jj=0;jj<CHANNUM*2;jj++)
		{
			hipfftExecR2C(Xplan,(hipfftReal *)&dev_x[jj*FILTER_FRAME],(hipfftComplex *)&dev_fft_x[jj*FILTER_FRAME]);
		}
		
		//Ƶ�����(13ms)
		DownSamplingFilter<<<CHANNUM*2*(FILTER_FRAME/2/THREADNUMPERBLK),THREADNUMPERBLK>>>(dev_fft_x,dev_fft_h,dev_fft_y,FILTER_FRAME);
	
		QueryPerformanceCounter(&nBeginTime); 
		//���任(105ms)
		for(int jj=0;jj<CHANNUM*2;jj++)
		{
			hipfftExecC2R(Yplan,(hipfftComplex *)&dev_fft_y[jj*FILTER_FRAME],(hipfftReal *)&dev_y[jj*FILTER_FRAME]);
			hipMemcpy(dev_chanbuff+jj*FILTER_FRAME/DOWNSAMPLE,dev_chanbuff+jj*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2,FILTER_FRAME/DOWNSAMPLE/2*sizeof(float),hipMemcpyDeviceToDevice);
		}
		IFFTNormalize<<<CHANNUM*2*(FILTER_FRAME/2/THREADNUMPERBLK),THREADNUMPERBLK>>>(dev_y,dev_chanbuff,FILTER_FRAME);	

		QueryPerformanceCounter(&nEndTime);
		//-----------------------------------------(1) �ź��˲�����������---------------------------------------------------


		//-----------------------------------------(2) Ƶ�����γ�---------------------------------------------------

		//ʹ�û������еĺ�FILTER_FRAME/DOWNSAMPLE/2��������Ƶ�����γɣ����Ʒ�λ
		for (int ii=0;ii<ARRAYNUM;ii++)		
		{		
			hipfftExecR2C(Beamplan,(hipfftReal *)&dev_chanbuff[ii*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2],(hipfftComplex *)&dev_fft[ii*(NFFT/2+1)]);
		}

		FD_Beamform<<<BlockRowNum*BEAMNUM,THREADNUMPERBLK>>>(dev_fft,dev_energy,PhiArray,nfl,nfh);//�����γ�
		MatrixSumRow<<<BEAMNUM,1>>>(dev_energy,sum_energy,BlockRowNum,BEAMNUM);
		
		hipMemcpy(c,sum_energy,BEAMNUM*sizeof(float),hipMemcpyDeviceToHost);
		fwrite(c,sizeof(float),BEAMNUM,fp);
		//-----------------------------------------(2) Ƶ�����γɽ���-----------------------------------------------


		//-----------------------------------------(3) �����������------------------------------------------
		//����������������
		memset(peak,0,BEAMNUM*sizeof(int));
		memset(valley,0,BEAMNUM*sizeof(int));
		findpeak(c,peak,BEAMNUM);
		findvalley(c,valley,BEAMNUM);
		bool targetexist = false;
		//memmove(beammatrix+BEAMNUM,beammatrix,4*BEAMNUM*sizeof(int));
		memset(pretracedtarget,0,sizeof(float)*BEAMNUM);
		memset(pretracedtargetIdx,0,sizeof(int)*BEAMNUM);
		pretracedtargetNum = 0;

		for(int kk=0;kk<BEAMNUM;kk++)
		{
			if(peak[kk] == 1)
			{
				//�ж��Ƿ��Ѹ��ٸò�������Ŀ��
				int jj=0;
				for(jj=0;jj<MAXTRACETARNUM;jj++)
				{
					//�����Ƿ��Ѹ���
					if(abs(tracedtargetbeam[jj][0]-kk)<6 && tracedtargetbeam[jj][0]>0)   //�Ѹ��ٸ�Ŀ�꣬���¸������Ƕ�
					{
						break;
					}
				}
				if(jj==MAXTRACETARNUM)  //δ����
				{
					targetexist = peakdetection(kk,c,valley,2.0);
				}
				else  //�Ѹ��٣����ͼ������
				{
					targetexist = peakdetection(kk,c,valley,1.2);
				}
				if(targetexist)
				{
					pretracedtarget[pretracedtargetNum] = c[kk];
					pretracedtargetIdx[pretracedtargetNum] = kk;
					pretracedtargetNum++;
				}
			}
		}
		rbub(pretracedtarget,pretracedtargetIdx,BEAMNUM);

		if(FrameNum == 115)
		{
			FrameNum = FrameNum;
		}
		for(int kk=0;kk<pretracedtargetNum;kk++)
		{
			int jj=0;
			for(jj=0;jj<MAXTRACETARNUM;jj++)
			{
				//�����Ƿ��Ѹ���
				if(abs(tracedtargetbeam[jj][0]-pretracedtargetIdx[kk])<6 && tracedtargetbeam[jj][0]>0)   //�Ѹ��ٸ�Ŀ�꣬���¸������Ƕ�
				{
					tracedtargetbeam[jj][0] = pretracedtargetIdx[kk];
					tracedtargetbeam[jj][1] = FrameNum;
					break;
				}
			}

			if(jj==MAXTRACETARNUM)  //δ���ٸ�Ŀ�꣬��һ���յĸ���������
			{
				int ii = 0;
				for(ii=0;ii<MAXTRACETARNUM;ii++)
				{
					//�����Ƿ��Ѹ���
					if(tracedtargetbeam[ii][0] < 0)
					{
						break;
					}
				}
				if(ii < MAXTRACETARNUM)           //�п��ø�����
				{
					tracedtargetbeam[ii][0] = pretracedtargetIdx[kk];
					tracedtargetbeam[ii][1] = FrameNum;
				}
			}
		}
		//������������ն�֡δ���µĸ�����
		for(int jj=0;jj<MAXTRACETARNUM;jj++)
		{
			if(tracedtargetbeam[jj][0] >0 && FrameNum - tracedtargetbeam[jj][1] >= 5)
			{
				tracedtargetbeam[jj][0] = -1;
				tracedtargetbeam[jj][1] = -1;
				tracedtargetangle[jj] = -1.0f;
			}
		}
		//-----------------------------------------(3) �����������-------------------------------------


		//-----------------------------------------(4) �������١����ٲ��� ------------------------------
	    hipMemset((void **)&dev_tracebeam_demon,0,sizeof(hipfftReal)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*MAXTRACETARNUM);
		for(int jj = 0;jj<MAXTRACETARNUM;jj++)
		{
			if(tracedtargetbeam[jj][0] >0)   //�и���Ŀ��
			{
				//�����ڲ�
				i0 = tracedtargetbeam[jj][0]-1;
				i1 = tracedtargetbeam[jj][0];
				i2 = tracedtargetbeam[jj][0]+1;
				r0 = c[i0];
				r1 = c[i1];
				r2 = c[i2];
				delta_index = (r2-r0)/(4*r1-2*r0-2*r2);
				tracedtargetangle[jj] = (i1+delta_index)*180.0/BEAMNUM;
				DelayFilterGen<<<ARRAYNUM,2*M+1>>>(dev_delayFilter,M,tracedtargetangle[jj],dev_tau,dev_dI);
				//DelayFilterGen<<<ARRAYNUM,2*M+1>>>(dev_delayFilter,M,60.292690,dev_tau,dev_dI);
				hipMemcpy(delayfiltercoff,dev_delayFilter,sizeof(hipfftReal)*ARRAYNUM*(2*M+1),hipMemcpyDeviceToHost);
				hipMemcpy(delaytau,dev_tau,sizeof(hipfftReal)*ARRAYNUM,hipMemcpyDeviceToHost);
				hipMemcpy(delaydI,dev_dI,sizeof(int)*ARRAYNUM,hipMemcpyDeviceToHost);
				
				for(int kk = 0;kk<ARRAYNUM;kk++)
				{
					if(delaydI[kk] >= 0)
					{
						hipMemcpy(dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE)+delaydI[kk],dev_chanbuff+kk*(FILTER_FRAME/DOWNSAMPLE),sizeof(hipfftReal)*((FILTER_FRAME/DOWNSAMPLE)-delaydI[kk]),hipMemcpyDeviceToDevice);
					}
					else
					{
						hipMemcpy(dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE),dev_chanbuff+kk*(FILTER_FRAME/DOWNSAMPLE)-delaydI[kk],sizeof(hipfftReal)*((FILTER_FRAME/DOWNSAMPLE)+delaydI[kk]),hipMemcpyDeviceToDevice);
					}

					//hipMemcpy(sourcedata,dev_chanbuff+kk*(FILTER_FRAME/DOWNSAMPLE),(FILTER_FRAME/DOWNSAMPLE)*sizeof(float),hipMemcpyDeviceToHost);
					//hipMemcpy(shiftdata,dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE),(FILTER_FRAME/DOWNSAMPLE)*sizeof(float),hipMemcpyDeviceToHost);

					if(fabs(delaytau[kk]) > 0.0001)
					{
						FineDelayFilter<<<(FILTER_FRAME/DOWNSAMPLE+2*M),2*M+1>>>((hipfftReal *)&dev_delayfilterbuf[kk*FILTER_FRAME/DOWNSAMPLE],(hipfftReal *)&dev_delayfilterout[kk*(FILTER_FRAME/DOWNSAMPLE+2*M)],(hipfftReal *)&dev_delayFilter[kk*(2*M+1)],M);
					}
					else
					{
						hipMemcpy(dev_delayfilterout+kk*(FILTER_FRAME/DOWNSAMPLE+2*M)+M,dev_delayfilterbuf+kk*(FILTER_FRAME/DOWNSAMPLE),sizeof(hipfftReal)*(FILTER_FRAME/DOWNSAMPLE),hipMemcpyDeviceToDevice);
					}
					hipMemcpy(dev_delaychandata+kk*(FILTER_FRAME/DOWNSAMPLE/2),dev_delayfilterout+kk*(FILTER_FRAME/DOWNSAMPLE+2*M)+M+FILTER_FRAME/DOWNSAMPLE/4,sizeof(hipfftReal)*FILTER_FRAME/DOWNSAMPLE/2,hipMemcpyDeviceToDevice);
				}
			
				MatrixSumRow<<<FILTER_FRAME/DOWNSAMPLE/2,1>>>(dev_delaychandata,dev_beamdata+jj*FILTER_FRAME/DOWNSAMPLE/2,ARRAYNUM,FILTER_FRAME/DOWNSAMPLE/2);
				hipMemcpy(beamdata,dev_beamdata+jj*FILTER_FRAME/DOWNSAMPLE/2,FILTER_FRAME/DOWNSAMPLE/2*sizeof(float),hipMemcpyDeviceToHost);
                //fwrite(beamdata,sizeof(float),FILTER_FRAME/DOWNSAMPLE/2,fpbeam);

				//������
                hipMemcpy(dev_tracebeam+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),dev_tracebeam+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)+(FILTER_FRAME/DOWNSAMPLE/2),(FILTER_FRAME/DOWNSAMPLE/2)*(PSD_LEN-1)*sizeof(hipfftReal),hipMemcpyDeviceToDevice);
                hipMemcpy(dev_tracebeam+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)+(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2),dev_beamdata+jj*FILTER_FRAME/DOWNSAMPLE/2,(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(hipfftReal),hipMemcpyDeviceToDevice);
				//�����׻�������λ
				hipMemcpy(dev_tracebeam_psd+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*PSD_AVG_NUM,dev_tracebeam_psd+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*PSD_AVG_NUM+PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),(PSD_AVG_NUM-1)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(hipfftReal),hipMemcpyDeviceToDevice);
                hipfftExecR2C(PSDplan,(hipfftReal *)&dev_tracebeam[jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)],(hipfftComplex *)&dev_tracebeam_spec[jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)]);
                Psd<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_tracebeam_spec+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),dev_tracebeam_psd+jj*PSD_LEN*PSD_AVG_NUM*(FILTER_FRAME/DOWNSAMPLE/2)+(PSD_AVG_NUM-1)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);
				//������ƽ��
				PsdAverage<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_tracebeam_psd+jj*PSD_LEN*PSD_AVG_NUM*(FILTER_FRAME/DOWNSAMPLE/2),dev_tracebeam_psd_avg+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
                hipMemcpy(trace_beam_psd,dev_tracebeam_psd_avg+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float),hipMemcpyDeviceToHost);
                //fwrite(trace_beam_psd,sizeof(float),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2,fpbeam);
				//������ƽ��
			    MySmooth(trace_beam_psd+idx1, idxLen, &fSmoothA[0][0], SMOOTH_N, 3, SMOOTH_N, 5, trace_beam_psd_smooth+idx1);
				MySmooth(trace_beam_psd_smooth+idx1, idxLen, &fSmoothA[0][0], SMOOTH_N, 2, SMOOTH_N, 5, trace_beam_psd_smooth+idx1);
				hipMemcpy(dev_tracebeam_psd_S,trace_beam_psd_smooth,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float),hipMemcpyHostToDevice);
				//�����ֵ��
				PsdSub<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_tracebeam_psd_avg+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),dev_tracebeam_psd_S,dev_tracebeam_psd_E,idx1,idx2);
				hipMemcpy(trace_beam_psd_smooth,dev_tracebeam_psd_E,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float),hipMemcpyDeviceToHost);
				//fwrite(trace_beam_psd_smooth,sizeof(float),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2,fpbeam);
				//�����ֵ�׷���
				fPsdEVar=0.0;
				for (int ii=idx1;ii<=idx2;ii++)
				{
					fPsdEVar+=trace_beam_psd_smooth[ii]*trace_beam_psd_smooth[ii];
				}
				fPsdEVar/=(float)(idx2-idx1+1);
				fPsdEVar=sqrtf(fPsdEVar);


				//�����
				for(int ii =0;ii<DemFreqBandNum;ii++)
				{
					hipMemcpy(dev_tracebeam_demonspec_band,dev_tracebeam_spec+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(hipfftComplex),hipMemcpyDeviceToDevice);
					//Ƶ���ͨ�˲�
					FrequencyDomainFilter<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_tracebeam_demonspec_band,fDf,DemStartFreq[ii],DemEndFreq[ii]);
					hipfftExecC2R(DEMONBandplan,dev_tracebeam_demonspec_band,dev_tracebeam_demon_band_data);
					SignalSqr<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_tracebeam_demon_band_data);				
					hipfftExecR2C(DEMONplan,dev_tracebeam_demon_band_data,dev_tracebeam_demonspec);
					DemonAdd<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_tracebeam_demonspec,dev_tracebeam_demon+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2), PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2));
				}
				hipMemcpy(trace_beam_demon,dev_tracebeam_demon+jj*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),DEM_RST_LEN*sizeof(float),hipMemcpyDeviceToHost);
				//ǰ�ĸ��㸳ֵ
				for(int ii=0;ii<6;ii++)
				{
					trace_beam_demon[ii] = trace_beam_demon[6];
				}
				//fwrite(trace_beam_demon,sizeof(float),DEM_RST_LEN,fpbeam);
				//�����ƽ��
			    MySmooth(trace_beam_demon, DEM_RST_LEN, &fSmoothA[0][0], SMOOTH_N, 3, SMOOTH_N, 5, trace_beam_demon_smooth);
				MySmooth(trace_beam_demon_smooth, DEM_RST_LEN, &fSmoothA[0][0], SMOOTH_N, 2, SMOOTH_N, 5, trace_beam_demon_smooth);
				//fwrite(trace_beam_demon_smooth,sizeof(float),DEM_RST_LEN,fpbeam);
				hipMemcpy(dev_trace_beam_demon_cut,trace_beam_demon,DEM_RST_LEN*sizeof(hipfftReal),hipMemcpyHostToDevice);
				hipMemcpy(dev_tracebeam_demon_S,trace_beam_demon_smooth,DEM_RST_LEN*sizeof(hipfftReal),hipMemcpyHostToDevice);
				//fwrite(trace_beam_demon_smooth,sizeof(float),DEM_RST_LEN,fpbeam)
				DemonSub<<<DEM_RST_LEN,1>>>(dev_trace_beam_demon_cut,dev_tracebeam_demon_S,dev_tracebeam_demon_E);
				hipMemcpy(trace_beam_demon_smooth,dev_tracebeam_demon_E,DEM_RST_LEN*sizeof(hipfftReal),hipMemcpyDeviceToHost);
				//fwrite(trace_beam_demon_smooth,sizeof(float),DEM_RST_LEN,fpbeam);
				fDemonEVar=0.0;
				for (int ii=0;ii<DEM_RST_LEN;ii++)
				{
					fDemonEVar+=trace_beam_demon_smooth[ii]*trace_beam_demon_smooth[ii];
				}
				fDemonEVar/=(float)(DEM_RST_LEN);
				fDemonEVar=sqrtf(fDemonEVar);

				//������ȡ
				int ll = 0;
				if(FrameNum >= 8)
				{
					nPlineNum = 0;
					memset(fPlineInfo,0,MAXTRACETARNUM*LINE_NUM*4*sizeof(float));
					for(int ii=idx1;ii<=idx2;ii++)
					{
						if(trace_beam_psd_smooth[ii]>4.0*fPsdEVar && trace_beam_psd_smooth[ii]>trace_beam_psd_smooth[ii-1] && trace_beam_psd_smooth[ii]>trace_beam_psd_smooth[ii+1] )
						{
							if(nPlineNum<LINE_NUM)
							{
								//���׹鲢
								for(ll = 0;ll<nPlineNum;ll++)
								{
									if(fabs(fPlineInfo[jj][ll][1]-(float)ii*fDf)<1.0)
									{
										break;
									}
								}
								if(ll == nPlineNum)
								{
									fPlineInfo[jj][nPlineNum][0] = trace_beam_psd_smooth[ii];    //�����
									fPlineInfo[jj][nPlineNum][1] = (float)ii*fDf;                //���������
									fPlineInfo[jj][nPlineNum][2] = trace_beam_psd[ii];
									fPlineInfo[jj][nPlineNum][3] = tracedtargetangle[jj];
									if(fPlineInfo[jj][nPlineNum][3] > 180.0)
									{
										fPlineInfo[jj][nPlineNum][3] -= 360.0;
									}
									else if(fPlineInfo[jj][nPlineNum][3] < -180.0)
									{
										fPlineInfo[jj][nPlineNum][3] += 360.0;
									}
									nPlineNum++;
								}
								else if(trace_beam_psd_smooth[ii] > fPlineInfo[jj][ll][0])
								{
									fPlineInfo[jj][ll][0] = trace_beam_psd_smooth[ii];
									fPlineInfo[jj][ll][1] = (float)ii*fDf;
									fPlineInfo[jj][ll][2] = trace_beam_psd[ii];
									fPlineInfo[jj][ll][3] = tracedtargetangle[jj];;
									if(fPlineInfo[jj][ll][3] > 180.0)
									{
										fPlineInfo[jj][ll][3] -= 360.0;
									}
									else if(fPlineInfo[jj][ll][3] < -180.0)
									{
										fPlineInfo[jj][ll][3] += 360.0;
									}		
								}
							}
						}
					}
					nDlineNum = 0;
					memset(fDlineInfo,0,MAXTRACETARNUM*LINE_NUM*2*sizeof(float));
					for(int ii = 4;ii<DEM_RST_LEN-1;ii++)
					{
						if(trace_beam_demon_smooth[ii]>6.0*fDemonEVar && trace_beam_demon_smooth[ii]>trace_beam_demon_smooth[ii-1] && trace_beam_demon_smooth[ii]>trace_beam_demon_smooth[ii+1])
						{
							if(nDlineNum<LINE_NUM)
							{
								fDlineInfo[jj][nDlineNum][0]=trace_beam_demon_smooth[jj];
								fDlineInfo[jj][nDlineNum][1]=ii*fDf;
								nDlineNum++;
							}
						}
					}
					//for(int ii = 0;ii<nDlineNum;ii++)
					//{
					//	printf("%d:%.3f\n",ii+1,fDlineInfo[jj][ii][1]);
					//}
				}
			}
		}

		//--------------------------ʸ������----------------------------------------------------
		hipMemcpy(dev_vector_p_buf,dev_vector_p_buf+(FILTER_FRAME/DOWNSAMPLE/2),(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float),hipMemcpyDeviceToDevice);
		hipMemcpy(dev_vector_p_buf+(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2),dev_chanbuff+VECTOR_P_IDX*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2,(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float),hipMemcpyDeviceToDevice);
		hipMemcpy(dev_vector_x_buf,dev_vector_x_buf+(FILTER_FRAME/DOWNSAMPLE/2),(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float),hipMemcpyDeviceToDevice);
		hipMemcpy(dev_vector_x_buf+(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2),dev_chanbuff+VECTOR_X_IDX*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2,(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float),hipMemcpyDeviceToDevice);
		hipMemcpy(dev_vector_y_buf,dev_vector_y_buf+(FILTER_FRAME/DOWNSAMPLE/2),(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float),hipMemcpyDeviceToDevice);
		hipMemcpy(dev_vector_y_buf+(PSD_LEN-1)*(FILTER_FRAME/DOWNSAMPLE/2),dev_chanbuff+VECTOR_Y_IDX*FILTER_FRAME/DOWNSAMPLE+FILTER_FRAME/DOWNSAMPLE/2,(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(float),hipMemcpyDeviceToDevice);

		hipfftExecR2C(PSDplan,(hipfftReal *)&dev_vector_p_buf[0],(hipfftComplex *)&dev_vector_p_spec[0]);
		hipfftExecR2C(PSDplan,(hipfftReal *)&dev_vector_x_buf[0],(hipfftComplex *)&dev_vector_x_spec[0]);
		hipfftExecR2C(PSDplan,(hipfftReal *)&dev_vector_y_buf[0],(hipfftComplex *)&dev_vector_y_spec[0]);

		hipMemcpy(dev_vector_p_psd,dev_vector_p_psd+PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),(PSD_AVG_NUM-1)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)*sizeof(hipfftReal),hipMemcpyDeviceToDevice);
        Psd<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_vector_p_spec,dev_vector_p_psd+(PSD_AVG_NUM-1)*PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2);
		PsdAverage<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_vector_p_psd,dev_vector_psd_avg);
		hipMemcpy(vector_p_psd,dev_vector_psd_avg,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float),hipMemcpyDeviceToHost);
		fwrite(vector_p_psd,sizeof(float),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2,fpbeam);

		MySmooth(vector_p_psd+idx1, idxLen, &fSmoothA[0][0], SMOOTH_N, 3, SMOOTH_N, 5, vector_p_psd_smooth+idx1);
		MySmooth(vector_p_psd_smooth+idx1, idxLen, &fSmoothA[0][0], SMOOTH_N, 2, SMOOTH_N, 5, vector_p_psd_smooth+idx1);
		hipMemcpy(dev_vector_p_psd_S,vector_p_psd_smooth,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float),hipMemcpyHostToDevice);
		PsdSub<<<PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2/THREADNUMPERBLK,THREADNUMPERBLK>>>(dev_vector_psd_avg,dev_vector_p_psd_S,dev_vector_p_psd_E,idx1,idx2);
		hipMemcpy(vector_p_psd_smooth,dev_vector_p_psd_E,PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2*sizeof(float),hipMemcpyDeviceToHost);
		//fwrite(vector_p_psd_smooth,sizeof(float),PSD_LEN*(FILTER_FRAME/DOWNSAMPLE/2)/2,fpbeam);
		fVectorPsdEVar=0.0;
		for (int ii=idx1;ii<=idx2;ii++)
		{
			fVectorPsdEVar+=vector_p_psd_smooth[ii]*vector_p_psd_smooth[ii];
		}
		fVectorPsdEVar/=(float)(idx2-idx1+1);
		fVectorPsdEVar=sqrtf(fVectorPsdEVar);
		//������ȡ
		int ll = 0;
		if(FrameNum >= 8)
		{
			nVectorPlineNum = 0;
			memset(fVectorPlineInfo,0,LINE_NUM*4*sizeof(float));
			for(int ii=idx1;ii<=idx2;ii++)
			{
				if(vector_p_psd_smooth[ii]>4.0*fVectorPsdEVar && vector_p_psd_smooth[ii]>vector_p_psd_smooth[ii-1] && vector_p_psd_smooth[ii]>vector_p_psd_smooth[ii+1] )
				{
					if(nVectorPlineNum<LINE_NUM)
					{
						//���׹鲢
						for(ll = 0;ll<nVectorPlineNum;ll++)
						{
							if(fabs(fVectorPlineInfo[ll][1]-(float)ii*fDf)<1.0)
							{
								break;
							}
						}
						if(ll == nVectorPlineNum)
						{
							fVectorPlineInfo[nVectorPlineNum][0] = vector_p_psd_smooth[ii];    //�����
							fVectorPlineInfo[nVectorPlineNum][1] = (float)ii*fDf;                //���������
							fVectorPlineInfo[nVectorPlineNum][2] = vector_p_psd[ii];
							//fVectorPlineInfo[nVectorPlineNum][3] = tracedtargetangle[jj];
							hipfftComplex P_f,Vx_f,Vy_f;
							hipMemcpy(&P_f,dev_vector_p_spec+ii,sizeof(hipfftComplex),hipMemcpyDeviceToHost);
							hipMemcpy(&Vx_f,dev_vector_x_spec+ii,sizeof(hipfftComplex),hipMemcpyDeviceToHost);
							hipMemcpy(&Vy_f,dev_vector_y_spec+ii,sizeof(hipfftComplex),hipMemcpyDeviceToHost);

							if(FrameNum == 20)
							{
								FrameNum = FrameNum;
							}

							fVectorPlineInfo[nVectorPlineNum][3] = VectorThetSPF(P_f, Vx_f, Vy_f);
							if(fVectorPlineInfo[nVectorPlineNum][3] > 180.0)
							{
								fVectorPlineInfo[nVectorPlineNum][3] -= 360.0;
							}
							else if(fVectorPlineInfo[nVectorPlineNum][3] < -180.0)
							{
								fVectorPlineInfo[nVectorPlineNum][3] += 360.0;
							}
							nVectorPlineNum++;
						}
						else if(vector_p_psd_smooth[ii] > fVectorPlineInfo[ll][0])
						{
							fVectorPlineInfo[ll][0] = vector_p_psd_smooth[ii];
							fVectorPlineInfo[ll][1] = (float)ii*fDf;
							fVectorPlineInfo[ll][2] = vector_p_psd[ii];
							hipfftComplex P_f,Vx_f,Vy_f;
							hipMemcpy(&P_f,dev_vector_p_spec+ii,sizeof(hipfftComplex),hipMemcpyDeviceToHost);
							hipMemcpy(&Vx_f,dev_vector_x_spec+ii,sizeof(hipfftComplex),hipMemcpyDeviceToHost);
							hipMemcpy(&Vy_f,dev_vector_y_spec+ii,sizeof(hipfftComplex),hipMemcpyDeviceToHost);

							fVectorPlineInfo[nVectorPlineNum][3] = VectorThetSPF(P_f, Vx_f, Vy_f);
							if(fVectorPlineInfo[ll][3] > 180.0)
							{
								fVectorPlineInfo[ll][3] -= 360.0;
							}
							else if(fVectorPlineInfo[ll][3] < -180.0)
							{
								fVectorPlineInfo[ll][3] += 360.0;
							}		
						}
					}
				}
			}
		}
		for(int ii = 0;ii<nVectorPlineNum;ii++)
		{
			printf("fVectorPlineInfo %d:%.3f\n",ii+1,fVectorPlineInfo[ii][3]);
		}
		//--------------------------------------------------------------------------------------
		hipEventRecord(stop1,NULL);
		hipEventSynchronize(stop1);
		//time=(double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart;
		hipEventElapsedTime(&msecTotal,start1,stop1);
		printf("%d:%f;%d,%d;%d,%d;%d,%d\n",FrameNum,msecTotal,tracedtargetbeam[0][0],tracedtargetbeam[0][1],tracedtargetbeam[1][0],tracedtargetbeam[1][1],tracedtargetbeam[2][0],tracedtargetbeam[2][1]);
		fprintf(fplog,"%d:%f;%d,%d;%d,%d;%d,%d\n",FrameNum,msecTotal,tracedtargetbeam[0][0],tracedtargetbeam[0][1],tracedtargetbeam[1][0],tracedtargetbeam[1][1],tracedtargetbeam[2][0],tracedtargetbeam[2][1]);
		fflush(fplog);
	}
	fclose(fp);
	fp = NULL;
	fclose(fplog);
	fplog = NULL;
	fclose(fpbeam);
	fpbeam = NULL;
}
int  fir1(int n,int band,float fl,float fh,float fs,int wn, float *h)
{
	int i,n2,mid;
	float sum = 0;
	float s,wc1,wc2,beta = 0,delay;
	float fln = fl / fs;
	float fhn = fh / fs;

	beta = 6;
	if((n%2)==0)
	{
		n2=n/2-1;
		mid=1;
	}
	else
	{
		n2=n/2;
		mid=0;
	}
	delay=n/2.0;
	wc1=2.0*PI*fln;
	if(band>=3) wc2=2.0*PI*fhn;
	switch(band)
	{
	case 1://��ͨ
		{
			for (i=0;i<=n2;i++)
			{
				s=i-delay;
				*(h+i)=(sin(wc1*s)/(PI*s))*window(wn,n+1,i,beta);
				*(h+n-i)=*(h+i);
			}
			if(mid==1) *(h+n/2)=wc1/PI;
			for(i=0;i<=n;i++)
			{
				sum=sum+*(h+i);
			}
			for(i=0;i<=n;i++)
			{
				*(h+i)=*(h+i)/fabs(sum);
			}
			break;
		}
	case 2: //��ͨ
		{
			for (i=0;i<=n2;i++)
			{
				s=i-delay;
				*(h+i)=(sin(PI*s)-sin(wc1*s))/(PI*s);
				*(h+i)=*(h+i)*window(wn,n+1,i,beta);
				*(h+n-i)=*(h+i);
			}
			if(mid==1) *(h+n/2)=1.0-wc1/PI;
			break;
		}
	case 3: //��ͨ
		{
			for (i=0;i<=n2;i++)
			{
				s=i-delay;
				*(h+i)=(sin(wc2*s)-sin(wc1*s))/(PI*s);
				*(h+i)=*(h+i)*window(wn,n+1,i,beta);
				*(h+n-i)=*(h+i);
			}
			if(mid==1) *(h+n/2)=(wc2-wc1)/PI;
			break;
		}
	case 4: //����
		{
			for (i=0;i<=n2;i++)
			{
				s=i-delay;
				*(h+i)=(sin(wc1*s)+sin(PI*s)-sin(wc2*s))/(PI*s);
				*(h+i)=*(h+i)*window(wn,n+1,i,beta);
				*(h+n-i)=*(h+i);
			}
			if(mid==1) *(h+n/2)=(wc1+PI-wc2)/PI;
			break;
		}
	}
	return 0;
}

float window(int type,int n,int i,float beta)
{
	int k;
	float w=1.0;
	switch(type)
	{
	case 1: //���δ�
		{
			w=1.0;
			break;
		}
	case 2: //ͼ����
		{
			k=(n-2)/10;
			if(i<=k) w=0.5*(1.0-cos(i*PI/(k+1)));
			if(i>n-k-2) w=0.5*(1.0-cos((n-i-1)*PI/(k+1)));
			break;
		}
	case 3: //���Ǵ�
		{
			w=1.0-fabs(1.0-2*i/(n-1.0));
			break;
		}
	case 4: //������
		{
			w=0.5*(1.0-cos(2*i*PI/(n-1.0)));
			break;
		}
	case 5: //������
		{
			w=0.54-0.46*cos(2*i*PI/(n-1.0));
			break;
		}
	case 6: //����������
		{
			w=0.42-0.5*cos(2*i*PI/(n-1.0))+0.08*cos(4*i*PI/(n-1.0));
			break;
		}
	case 7: //������
		{
			w=kaiser(i,n,beta);
			break;
		}
	}
	return(w);
}

float kaiser(int i,int n,float beta)  //��������iΪ��ţ�nΪ�˲�������
{
	float a,w,a2,b1,b2,beta1;
	b1=bessel0(beta);
	a=2.0*i/(float)(n-1)-1.0;
	a2=a*a;
	beta1=beta*sqrt(1.0-a2);
	b2=bessel0(beta1);
	w=b2/b1;
	return(w);
}

float bessel0(float x)  //��ױ���������
{
	int i;
	float dd,y,d2,sum = 0;
	y=x/2.0;
	dd=1.0;
	for(i=1;i<=25;i++)
	{
		dd=dd*y/i;
		d2=dd*dd;
		sum=sum+d2;
		if(d2<sum*(1.0e-8)) break;
	}
	return(sum);
}

void findpeak(float *data, int *p,int dn)
{
	int acc=0,acc1=0;
	int i,j;
	float a0=0.0,a1=0.0;
	for(i=0;i<dn;i++)
	{
		a0=*(data+i);
		//����ǰ��
        for(j=1;j<11;j++)
		{
			if ((i+j)>=dn)
			{
				a1=*(data+i+j-dn);
			}
			else
			{
				a1=*(data+i+j);
			}
			if (a0>a1)
			{
				acc=acc+1;
			}
		}
        a0=*(data+i);
		////�������
        for(j=1;j<11;j++)
		{
			if ((i-j)<0)
			{
				a1=*(data+i-j+dn);
			}
			else
			{
				a1=*(data+i-j);
			}
			if (a0>a1)
			{
				acc1=acc1+1;
			}
		}
	  if ((acc==10) && (acc1==10))
	  {
         *(p+i)=1;
	  }
	  acc=0;
      acc1=0;
	}
}

void findvalley(float *data, int *p,int dn)
{
	int acc=0,acc1=0;
	int i,j;
	float a0=0.0,a1=0.0;
	for(i=0;i<dn;i++)
	{
		a0=*(data+i);
		//����ǰ��
        for(j=1;j<6;j++)
		{
			if ((i+j)>=dn)
			{
				//a1=*(data+i+j-dn);
				break;
			}
			else
			{
				a1=*(data+i+j);
			}
			if (a0<a1)
			{
				acc=acc+1;
			}
		}
		if(j<5)  //ѭ����break�˳������˲��������ֵ
		{
			acc = 5; 
		}
        a0=*(data+i);
		////�������
        for(j=1;j<6;j++)
		{
			if ((i-j)<0)
			{
				//a1=*(data+i-j+dn);
				break;
			}
			else
			{
				a1=*(data+i-j);
			}
			if (a0<a1)
			{
				acc1=acc1+1;
			}
		}
		if(j<5)  //ѭ����break�˳������˲�������Сֵ
		{
			acc1 = 5; 
		}
		if ((acc==5) && (acc1==5))
		{
		    *(p+i)=1;
		}
		acc=0;
		acc1=0;
	}
}

bool peakdetection(int beamidx,float *be,int *valley,float threshold)
{
	int index = 0,ll=0;
	float pvr1 = 1.0,pvr2 = 1.0;
	if(beamidx >= STARTBEAM && beamidx <= ENDBEAM)
	{
		for(ll=beamidx+1;ll<BEAMNUM;ll++)
		{
			if(valley[ll] == 1)
			{
				index = ll;
				break;
			}
		}
		if(ll<=BEAMNUM-1)
		{
			pvr1 = be[beamidx] / be[index];
		}

		for(ll=beamidx-1;ll>=0;ll--)
		{
			if(valley[ll] == 1)
			{
				index = ll;
				break;
			}
		}
		if(ll>=0)
		{
			pvr2 = be[beamidx] / be[index];
		}

		if(pvr1 >= threshold && pvr2 >= threshold)
		{
			return true;
		}
		else
		{
			return false;
		}
	}
	else
	{
		return false;
	}
}

void rbub(float *p,int *idx,int n)
{ 
	int m,k,j,i,xx;
    float dd;
    
	k=0; 
	m=n-1;
    while (k<m)
    { 
		j=m-1; m=0;
        for(i=k; i<=j; i++)
		{
			if(p[i]<p[i+1])
			{ 
				dd=p[i]; 
				p[i]=p[i+1]; 
				p[i+1]=dd; 
				xx = idx[i];
				idx[i] = idx[i+1];
				idx[i+1] = xx;
				m=i;
			}
		}
        j=k+1; 
		k=0;
        for (i=m; i>=j; i--)
		{
			if(p[i-1]<p[i])
			{ 
				dd=p[i]; 
				p[i]=p[i-1]; 
				p[i-1]=d; 
				xx = idx[i];
				idx[i] = idx[i-1];
				idx[i-1] = xx;
				k=i;
			}
		}
      }
    return;
  }

void MySmooth(float *datain,int nDataLen,float *paraA,int nParaLen,int nOrder,int nWindow,int nStep,float *dataout)
{
	int nFrameNum,ii,jj,nFrameCnt,idx;
	float rr[4]={0};
	float fsmooth_tmp[SMOOTH_N]={0};
	float fsmooth_tmp2[SMOOTH_N]={0};

	nFrameNum=(nDataLen-nWindow)/nStep+1;

	for (nFrameCnt=0;nFrameCnt<nFrameNum;nFrameCnt++)
	{
		if(nFrameCnt==0)
		{
			memcpy(fsmooth_tmp,datain,nWindow*sizeof(float));
		}
		else
		{
			memcpy(&fsmooth_tmp[nWindow-nStep],&datain[nWindow+(nFrameCnt-1)*nStep],nStep*sizeof(float));
		}

		for (ii=0;ii<nOrder;ii++)
		{
			rr[ii]=0.0;
			for (jj=0;jj<nWindow;jj++)
			{
				rr[ii]+=fsmooth_tmp[jj]*fSmoothA[ii][jj];
			}
		}

		memset(fsmooth_tmp2,0,SMOOTH_N*sizeof(float));
		for (ii=0;ii<nWindow;ii++)
		{
				for (jj=0;jj<nOrder;jj++)
				{
					fsmooth_tmp2[ii]+=rr[jj]*fSmoothA[jj][ii];
				}
		}

		memcpy(&dataout[nFrameCnt*nStep],fsmooth_tmp2,nStep*sizeof(float));
		memcpy(fsmooth_tmp,&fsmooth_tmp2[nStep],(nWindow-nStep)*sizeof(float));
	}//for (nFrameCnt=0;nFrameCnt<nFrameNum-1;nFrameCnt++)

	if ((nFrameNum*nStep+nWindow)-nDataLen<nStep)
	{
		idx=(nFrameNum*nStep+nWindow)-nDataLen;
		memcpy(fsmooth_tmp,&fsmooth_tmp2[nStep-idx],(nWindow-nStep+idx)*sizeof(float));
		memcpy(&fsmooth_tmp[nWindow-nStep+idx],&datain[nWindow+(nFrameNum-1)*nStep],(nStep-idx)*sizeof(float));

		for (ii=0;ii<nOrder;ii++)
		{
			rr[ii]=0.0;
			for (jj=0;jj<nWindow;jj++)
			{
				rr[ii]+=fsmooth_tmp[jj]*fSmoothA[ii][jj];
			}
		}

		memset(fsmooth_tmp2,0,SMOOTH_N*sizeof(float));
		for (ii=0;ii<nWindow;ii++)
		{
			for (jj=0;jj<nOrder;jj++)
			{
				fsmooth_tmp2[ii]+=rr[jj]*fSmoothA[jj][ii];
			}
		}

		memcpy(&dataout[nFrameNum*nStep],&fsmooth_tmp2[idx],(nWindow-idx)*sizeof(float));

	}
	else//if ((nFrameNum*nStep+nWindow)-nDataLen<nStep)
	{
		memcpy(&dataout[nFrameNum*nStep],&fsmooth_tmp2[nStep],(nWindow-nStep)*sizeof(float));

	}//if ((nFrameNum*nStep+nWindow)-nDataLen<nStep)
}

void CalSmoothPara(float *para)
{
	float fpara[4][SMOOTH_N];
	float ftmp,ftmp2,ftmp3;
	int ii,jj;

	ftmp=sqrtf((float)(SMOOTH_N));
	ftmp=1.0/ftmp;
	for (ii=0;ii<SMOOTH_N;ii++)
	{
		fpara[0][ii]=ftmp;
	}

	ftmp2=0;

	for (ii=0;ii<SMOOTH_N;ii++)
	{
		fpara[1][ii]=(float)(ii-(SMOOTH_N-1)/2);
		fpara[2][ii]=fpara[1][ii]*fpara[1][ii];
		ftmp2+=fpara[2][ii];
		fpara[3][ii]=fpara[2][ii]*fpara[1][ii];
	}
	ftmp=1.0/sqrtf(ftmp2);
	ftmp3=0;
	for (ii=0;ii<SMOOTH_N;ii++)
	{
		fpara[1][ii]=fpara[1][ii]*ftmp;
		ftmp3+=fpara[1][ii]*fpara[3][ii];
	}

	ftmp=0;
	ftmp2=ftmp2/(float)(SMOOTH_N);
	for (ii=0;ii<SMOOTH_N;ii++)
	{
		fpara[2][ii]=fpara[2][ii]-ftmp2;
		ftmp+=fpara[2][ii]*fpara[2][ii];
	}
	ftmp=1.0/sqrtf(ftmp);
	ftmp2=0;
	for (ii=0;ii<SMOOTH_N;ii++)
	{
		fpara[2][ii]=fpara[2][ii]*ftmp;
		fpara[3][ii]=fpara[3][ii]-ftmp3*fpara[1][ii];
		ftmp2+=fpara[3][ii]*fpara[3][ii];
	}
	ftmp=1.0/sqrtf(ftmp2);
	for (ii=0;ii<SMOOTH_N;ii++)
	{
		fpara[3][ii]=fpara[3][ii]*ftmp;
	}

	memcpy(para,&fpara[0][0],sizeof(float)*4*SMOOTH_N);
}
